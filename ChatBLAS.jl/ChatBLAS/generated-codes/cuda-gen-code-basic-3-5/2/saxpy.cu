#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__
void saxpy(int n, float a, float *x, float *y) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        y[i] = a * x[i] + y[i];
    }
}

void chatblas_saxpy(int n, float a, float *x, float *y) {
    float *d_x, *d_y;
    int size = n * sizeof(float);
  
    // Allocate device memory
    hipMalloc((void **)&d_x, size);
    hipMalloc((void **)&d_y, size);

    // Copy input vectors from host memory to device memory
    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

    // Calculate block and grid sizes
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the SAXPY kernel
    saxpy<<<blocksPerGrid, threadsPerBlock>>>(n, a, d_x, d_y);

    // Copy result back to host memory
    hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_x);
    hipFree(d_y);
}
