#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void scaleVector(int n, float a, float *x) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        x[idx] *= a;
    }
}

void chatblas_sscal(int n, float a, float *x) {
    float *dev_x;
    if (hipMalloc((void **)&dev_x, n * sizeof(float)) != hipSuccess) {
        printf("Failed to allocate device memory for x.\n");
        return;
    }

    if (hipMemcpy(dev_x, x, n * sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
        printf("Failed to copy x to device memory.\n");
        hipFree(dev_x);
        return;
    }

    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;

    scaleVector<<<gridSize, blockSize>>>(n, a, dev_x);

    if (hipMemcpy(x, dev_x, n * sizeof(float), hipMemcpyDeviceToHost) != hipSuccess) {
        printf("Failed to copy scaled x back to host memory.\n");
    }

    hipFree(dev_x);
}
