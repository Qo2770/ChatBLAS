#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sasum_kernel(int n, float *x, float *result) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n) {
        // Compute absolute value and add to the result
        atomicAdd(result, fabsf(x[tid]));
    }
}

float chatblas_sasum(int n, float *x) {
    // Allocate memory on the GPU for x and the result
    float *d_x, h_result, *d_result;
    hipMalloc((void **)&d_x, n * sizeof(float));
    hipMalloc((void **)&d_result, sizeof(float));

    // Copy input vector x from host to the GPU memory
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);

    // Set the initial value of the result to 0 on the GPU
    hipMemcpy(d_result, &h_result, sizeof(float), hipMemcpyHostToDevice);

    // Number of threads per block
    int threadsPerBlock = 256;

    // Number of blocks needed
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel
    sasum_kernel<<<blocksPerGrid, threadsPerBlock>>>(n, d_x, d_result);

    // Copy the result from GPU memory to host
    hipMemcpy(&h_result, d_result, sizeof(float), hipMemcpyDeviceToHost);
    
    // Free the GPU memory
    hipFree(d_x);
    hipFree(d_result);

    return h_result;
}
