#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__
void saxpy(int n, float a, float *x, float *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        y[i] = a * x[i] + y[i];
    }
}

void chatblas_saxpy(int n, float a, float *x, float *y) {
    float *d_x, *d_y;

    // Allocate device memory
    hipMalloc((void **)&d_x, n * sizeof(float));
    hipMalloc((void **)&d_y, n * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);

    // Define grid and block sizes for CUDA kernel execution
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;

    // Launch the CUDA kernel
    saxpy<<<gridSize, blockSize>>>(n, a, d_x, d_y);

    // Copy the result back from device to host
    hipMemcpy(y, d_y, n * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_x);
    hipFree(d_y);
}
