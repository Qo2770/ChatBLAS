#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

void chatblas_scopy(int n, float *x, float *y) {
    // Allocate memory on the GPU
    float *d_x, *d_y;
    hipMalloc((void **)&d_x, sizeof(float) * n);
    hipMalloc((void **)&d_y, sizeof(float) * n);

    // Copy input vectors from host memory to GPU buffers
    hipMemcpy(d_x, x, sizeof(float) * n, hipMemcpyHostToDevice);

    // Launch the copy kernel
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;
    copyKernel<<<gridSize, blockSize>>>(n, d_x, d_y);

    // Copy output vector from GPU buffer to host memory
    hipMemcpy(y, d_y, sizeof(float) * n, hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_x);
    hipFree(d_y);
}

__global__ void copyKernel(int n, float *x, float *y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n)
        y[index] = x[index];
}
