#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void dotProduct(int n, float *x, float *y, float *result) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    
    float localSum = 0.0f;
    for (int i = index; i < n; i += stride) {
        localSum += x[i] * y[i];
    }
    
    atomicAdd(result, localSum);
}

float chatblas_sdot(int n, float *x, float *y) {
    float *dev_x, *dev_y, *dev_result;
    
    // Allocate memory on the device
    hipMalloc((void**)&dev_x, n * sizeof(float));
    hipMalloc((void**)&dev_y, n * sizeof(float));
    hipMalloc((void**)&dev_result, sizeof(float));
    
    // Copy input vectors from host to device memory
    hipMemcpy(dev_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_y, y, n * sizeof(float), hipMemcpyHostToDevice);
    
    // Set initial value of the result on the device to 0
    hipMemset(dev_result, 0, sizeof(float));
    
    // Launch kernel to compute the dot product in parallel
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    dotProduct<<<numBlocks, blockSize>>>(n, dev_x, dev_y, dev_result);
    
    // Copy the result back from the device to the host
    float result;
    hipMemcpy(&result, dev_result, sizeof(float), hipMemcpyDeviceToHost);
    
    // Free device memory
    hipFree(dev_x);
    hipFree(dev_y);
    hipFree(dev_result);
    
    return result;
}
