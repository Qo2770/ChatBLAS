#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"
#include <hip/hip_runtime.h>

__global__ void axpyKernel(int n, float a, float *x, float *y) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        y[i] = a * x[i] + y[i];
    }
}

void chatblas_saxpy(int n, float a, float *x, float *y) {
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    
    float *d_x, *d_y;
    hipMalloc(&d_x, n * sizeof(float));
    hipMalloc(&d_y, n * sizeof(float));

    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);

    axpyKernel<<<numBlocks, blockSize>>>(n, a, d_x, d_y);

    hipMemcpy(y, d_y, n * sizeof(float), hipMemcpyDeviceToHost);
  
    hipFree(d_x);
    hipFree(d_y);
}
