#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

// CUDA Kernel that calculates dot product
__global__ void dotProductKernel(float *x, float *y, float *dot, int n){
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for(int i=index; i<n; i+=stride)
        dot[index] += ((double)x[i]) * ((double)y[i]);
}

// Sum reduction function
__device__ void reduction(float *input, int n){
    int threadId = threadIdx.x;
    int blockSize = blockDim.x;
	
	int i = blockSize / 2;
	while(i != 0){
		if(threadId < i){
			input[threadId] += input[threadId + i];
		}
		__syncthreads();
		i /= 2;
	}
}

float chatblas_sdsdot(int n, float b, float *x, float *y){

    int size = n*sizeof(float);
    float *dot;
    float *d_x, *d_y, *d_dot;

    dot = (float*)malloc(size);

    hipMalloc(&d_x, size);
    hipMalloc(&d_y, size);
    hipMalloc(&d_dot, size);

    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

    int blockSize = 1024;
    int numBlocks = (n + blockSize - 1) / blockSize;
    dotProductKernel<<<numBlocks, blockSize>>>(d_x, d_y, d_dot, n);
	
    reduction<<<1, blockSize>>>(d_dot, n);
	
    hipMemcpy(dot, d_dot, size, hipMemcpyDeviceToHost);
	
    float result = dot[0] + b;

    free(dot);
    hipFree(d_x); hipFree(d_y); hipFree(d_dot);

    return result;
}
