#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"
#include "hip/hip_runtime.h"
#include ""

__global__ void chatblas_saxpy_kernel(int n, float a, float *x, float *y)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) 
    {
        y[i] = a * x[i] + y[i];
    }
}

void chatblas_saxpy(int n, float a, float *x, float *y)
{
    int threadsPerBlock = 256;
    int blocksPerGrid =(n + threadsPerBlock - 1) / threadsPerBlock;
    chatblas_saxpy_kernel<<<blocksPerGrid, threadsPerBlock>>>(n, a, x, y);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
    {
        printf("Error: %s\n", hipGetErrorString(err));
    }
}
