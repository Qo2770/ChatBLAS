#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

// Kernel function to copy contents of one array to another
__global__ void copy_kernel(int n, float *x, float *y)
{
    // Get the thread ID which equals the vector index
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) {
        y[i] = x[i];
    }
}

void chatblas_scopy(int n, float *x, float *y) 
{
   int blockSize = 1024;
   int numBlocks = (n + blockSize - 1) / blockSize;

   // Allocate device memory
   float *d_x, *d_y;
   hipMalloc((void **)&d_x, n * sizeof(float));
   hipMalloc((void **)&d_y, n * sizeof(float));

   // Copy vector x from host to device memory
   hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);

   //call kernel function 
   copy_kernel<<<numBlocks, blockSize>>>(n, d_x, d_y);

   // Copy the result back to host memory
   hipMemcpy(y, d_y, n * sizeof(float), hipMemcpyDeviceToHost);

   // Free device memory
   hipFree(d_x); hipFree(d_y);
}
