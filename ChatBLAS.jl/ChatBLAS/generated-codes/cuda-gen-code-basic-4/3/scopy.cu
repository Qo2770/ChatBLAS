#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void chatblas_scopy_cuda(int n, float *x, float *y) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        y[index] = x[index];
    }
}

void chatblas_scopy(int n, float *x, float *y) {
    float* d_x;
    float* d_y;

    hipMalloc(&d_x, n*sizeof(float)); 
    hipMalloc(&d_y, n*sizeof(float));

    hipMemcpy(d_x, x, n*sizeof(float), hipMemcpyHostToDevice);

    chatblas_scopy_cuda<<<(n+255)/256, 256>>>(n, d_x, d_y);

    hipMemcpy(y, d_y, n*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
}
