#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void dotProdKernel(int n, float *x, float *y, float *result) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    __shared__ float temp[256];
    temp[threadIdx.x] = 0;

    if(index < n) {
        temp[threadIdx.x] = x[index] * y[index];
    }
    __syncthreads();

    if(threadIdx.x == 0) {
        float sum = 0;
        for(int i=0; i<256; i++) {
            sum += temp[i];
        }
        atomicAdd(result, sum);
    }
}

float chatblas_sdot(int n, float *x, float *y) {
    float *d_x, *d_y, *d_result;
    float result = 0;
    int size = n * sizeof(float); 

    hipMalloc(&d_x, size);
    hipMalloc(&d_y, size);
    hipMalloc(&d_result, sizeof(float));

    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

    dotProdKernel<<<(n+255)/256, 256>>>(n, d_x, d_y, d_result);

    hipMemcpy(&result, d_result, sizeof(float), hipMemcpyDeviceToHost); // Result copied from device to host

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_result);
    
    return result;
}
