#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

// CUDA Kernel Function
__global__ void dot_product(int n, float b, float *x, float *y, float *result) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        result[tid] = ((double)x[tid] * (double)y[tid]) + (double)b;
    }
}

// Main Function
float chatblas_sdsdot(int n, float b, float *x, float *y) {
    float *d_x, *d_y, *d_result;

    // Allocate memory on the device
    hipMalloc((void**)&d_x, n * sizeof(float));
    hipMalloc((void**)&d_y, n * sizeof(float));
    hipMalloc((void**)&d_result, n * sizeof(float));

    // Copy vectors from host to device
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);
    
    // Set grid and block dimensions
    dim3 blockDim(256);
    dim3 gridDim((n + blockDim.x - 1) / blockDim.x);
    
    // Launch CUDA kernel
    dot_product<<<gridDim, blockDim>>>(n, b, d_x, d_y, d_result);
    
    float *h_result = (float*) malloc(n * sizeof(float));
    // Copy the result from device to host
    hipMemcpy(h_result, d_result, n * sizeof(float), hipMemcpyDeviceToHost);

    // Sum the results
    float sum = 0;
    for (int i = 0; i < n; ++i) {
        sum += h_result[i];
    }

    // Free the device memory
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_result);

    // Free the host memory
    free(h_result);

    return sum;
}
