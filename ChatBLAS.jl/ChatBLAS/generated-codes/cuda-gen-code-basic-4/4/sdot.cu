#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"
#include <hipblas.h>

__global__ void dotProdKernel(int n, float *x, float *y, float *result) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    __shared__ float temp[MAX_THREADS_PER_BLOCK];

    temp[threadIdx.x] = 0;

    if (index < n)
        temp[threadIdx.x] = x[index] * y[index];

    __syncthreads();

    if (threadIdx.x == 0) {
        float sum = 0;
        for (int i = 0; i < blockDim.x; i++)
            sum += temp[i];

        atomicAdd(result, sum);
    }
}

float chatblas_sdot(int n, float *x, float *y) {
    float *dev_x, *dev_y, *dev_result;
    float result;

    hipMalloc((void**)&dev_x, n * sizeof(float));
    hipMalloc((void**)&dev_y, n * sizeof(float));
    hipMalloc((void**)&dev_result, sizeof(float));

    hipMemcpy(dev_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_y, y, n * sizeof(float), hipMemcpyHostToDevice);

    result = 0.0;
    hipMemcpy(dev_result, &result, sizeof(float), hipMemcpyHostToDevice);

    dotProdKernel<<<(n + MAX_THREADS_PER_BLOCK - 1) / MAX_THREADS_PER_BLOCK, MAX_THREADS_PER_BLOCK>>>(n, dev_x, dev_y, dev_result);

    hipMemcpy(&result, dev_result, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(dev_x);
    hipFree(dev_y);
    hipFree(dev_result);

    return result;
}
