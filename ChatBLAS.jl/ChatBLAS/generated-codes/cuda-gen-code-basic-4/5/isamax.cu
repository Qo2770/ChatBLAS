#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"
#include <hipblas.h>

__global__ void abs_max_index_kernel(float* input, int* output, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < n && abs(input[idx]) > abs(input[*output]))
        atomicExch(output, idx);
}

int chatblas_isamax(int n, float *x) {
    float *dev_x;
    int *dev_output;
    
    hipMalloc((void** ) &dev_x, n * sizeof(float));
    hipMemcpy(dev_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    
    // initial index
    int initial = 0;
    hipMalloc((void** ) &dev_output, sizeof(int));
    hipMemcpy(dev_output, &initial, sizeof(int), hipMemcpyHostToDevice);
    
    // launch the kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(n + threadsPerBlock - 1) / threadsPerBlock;
    abs_max_index_kernel<<<blocksPerGrid, threadsPerBlock>>>(dev_x, dev_output, n);

    int output;
    hipMemcpy(&output, dev_output, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_x);
    hipFree(dev_output);
    
    return output;
}
