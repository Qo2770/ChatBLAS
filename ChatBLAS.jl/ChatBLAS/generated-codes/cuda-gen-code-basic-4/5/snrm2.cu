#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"
#include <math.h>

__global__ void vectorNormKernel(int n, float *d_x, float *d_result)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < n)
    {
        float element = d_x[idx];
        atomicAdd(d_result, element * element);
    }
}

float chatblas_snrm2(int n, float *x)
{
    float *d_x, *d_result;
    float result = 0;

    hipMalloc((void **) &d_x, n*sizeof(float));
    hipMemcpy(d_x, x, n*sizeof(float), hipMemcpyHostToDevice);

    hipMalloc((void **) &d_result, sizeof(float));
    hipMemcpy(d_result, &result, sizeof(float), hipMemcpyHostToDevice);

    vectorNormKernel<<<(n+255)/256, 256>>>(n, d_x, d_result);
    
    hipMemcpy(&result, d_result, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_result);

    return sqrt(result);
}
