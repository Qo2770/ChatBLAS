#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void saxpy_kernel(int n, float a, float *x, float *y) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        y[i] += a * x[i];
    }
}

void chatblas_saxpy(int n, float a, float *x, float *y) {
    float *d_x, *d_y;
    size_t size = n * sizeof(float);

    hipMalloc(&d_x, size);
    hipMalloc(&d_y, size);

    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    saxpy_kernel<<<numBlocks, blockSize>>>(n, a, d_x, d_y);

    hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
}