#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA kernel to copy elements from vector x to vector y
__global__ void scopy_kernel(int n, const float *x, float *y) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        y[idx] = x[idx];
    }
}

// Function to copy vector x to vector y using CUDA
void chatblas_scopy(int n, float *x, float *y) {
    // Allocate device memory
    float *d_x, *d_y;
    hipMalloc((void**)&d_x, n * sizeof(float));
    hipMalloc((void**)&d_y, n * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);

    // Define block size and compute number of blocks
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    // Launch the CUDA kernel
    scopy_kernel<<<numBlocks, blockSize>>>(n, d_x, d_y);

    // Copy the result from device to host
    hipMemcpy(y, d_y, n * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_x);
    hipFree(d_y);
}
