#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void saxpy_kernel(int n, float a, float *x, float *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        y[i] += a * x[i];
    }
}

void chatblas_saxpy(int n, float a, float *x, float *y) {
    float *d_x, *d_y;
    int size = n * sizeof(float);

    hipMalloc((void **) &d_x, size);
    hipMalloc((void **) &d_y, size);

    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    saxpy_kernel<<<numBlocks, blockSize>>>(n, a, d_x, d_y);

    hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
}
