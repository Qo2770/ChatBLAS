#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"
#include <hip/hip_runtime.h>

// CUDA kernel to scale a vector
__global__ void scaleKernel(int n, float a, float *x) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        x[idx] *= a;
    }
}

// Host function to scale a vector using CUDA
void chatblas_sscal(int n, float a, float *x) {
    if (n <= 0) return;

    // Choose a reasonable number of threads per block
    int threadsPerBlock = 256;
    int blocks = (n + threadsPerBlock - 1) / threadsPerBlock; // Calculate number of blocks needed

    // Allocate memory on the GPU
    float *d_x;
    size_t size = n * sizeof(float);
    hipMalloc((void **)&d_x, size);

    // Copy vector from host to GPU
    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);

    // Launch kernel
    scaleKernel<<<blocks, threadsPerBlock>>>(n, a, d_x);

    // Copy result back to host
    hipMemcpy(x, d_x, size, hipMemcpyDeviceToHost);

    // Free allocated memory on the GPU
    hipFree(d_x);
}
