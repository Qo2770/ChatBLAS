#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

// CUDA kernel to copy vector x to vector y
__global__ void copyKernel(int n, const float *x, float *y) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        y[idx] = x[idx];
    }
}

// Function to copy a vector x to vector y using CUDA
void chatblas_scopy(int n, float *x, float *y) {
    // Define CUDA grid and block sizes
    const int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    // Allocate device memory
    float *d_x, *d_y;
    hipMalloc((void**)&d_x, n * sizeof(float));
    hipMalloc((void**)&d_y, n * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);

    // Launch the kernel to perform the copy
    copyKernel<<<numBlocks, blockSize>>>(n, d_x, d_y);

    // Copy result from device to host
    hipMemcpy(y, d_y, n * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_x);
    hipFree(d_y);
}
