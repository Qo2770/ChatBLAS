#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void dotProductKernel(float *x, float *y, float *result, int n) {
    extern __shared__ float temp[];

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int threadId = threadIdx.x;

    // Load elements into shared memory and compute the partial dot product
    temp[threadId] = (index < n) ? x[index] * y[index] : 0.0f;
    __syncthreads();

    // Perform parallel reduction
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (threadId < stride)
            temp[threadId] += temp[threadId + stride];
        __syncthreads();
    }

    // Thread 0 writes the result for this block to global memory
    if (threadId == 0)
        atomicAdd(result, temp[0]);
}

float chatblas_sdot(int n, float *x, float *y) {
    float *d_x, *d_y, *d_result;
    float h_result = 0.0f;

    hipMalloc((void**)&d_x, n * sizeof(float));
    hipMalloc((void**)&d_y, n * sizeof(float));
    hipMalloc((void**)&d_result, sizeof(float));

    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_result, &h_result, sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;
    dotProductKernel<<<gridSize, blockSize, blockSize * sizeof(float)>>>(d_x, d_y, d_result, n);

    hipMemcpy(&h_result, d_result, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_result);

    return h_result;
}
