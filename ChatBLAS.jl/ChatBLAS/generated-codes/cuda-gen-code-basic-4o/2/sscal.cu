#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA kernel to scale a vector
__global__ void scale_vector(int n, float a, float *x) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        x[i] *= a;
    }
}

void chatblas_sscal(int n, float a, float *x) {
    // Define block size for CUDA kernel
    const int blockSize = 256;
    const int numBlocks = (n + blockSize - 1) / blockSize;

    // Allocate memory on the GPU
    float *d_x;
    hipMalloc((void **)&d_x, n * sizeof(float));

    // Copy the input vector from host to device
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);

    // Launch the CUDA kernel
    scale_vector<<<numBlocks, blockSize>>>(n, a, d_x);

    // Copy the result back to the host
    hipMemcpy(x, d_x, n * sizeof(float), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_x);
}
