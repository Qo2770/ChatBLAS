#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"
#include <hip/hip_runtime.h>

__global__ void vectorCopyKernel(int n, const float *x, float *y) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        y[idx] = x[idx];
    }
}

void chatblas_scopy(int n, float *x, float *y) {
    // Define device pointers
    float *d_x, *d_y;

    // Allocate memory on the device
    hipMalloc((void**)&d_x, n * sizeof(float));
    hipMalloc((void**)&d_y, n * sizeof(float));

    // Copy input vector from host to device
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);

    // Define block and grid sizes
    int blockSize = 256; // You can choose a suitable block size based on your hardware
    int gridSize = (n + blockSize - 1) / blockSize;

    // Launch the kernel
    vectorCopyKernel<<<gridSize, blockSize>>>(n, d_x, d_y);

    // Copy result back to host
    hipMemcpy(y, d_y, n * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_x);
    hipFree(d_y);
}
