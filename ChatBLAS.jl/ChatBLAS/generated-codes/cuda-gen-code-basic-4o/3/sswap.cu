#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void swap_kernel(int n, float *x, float *y) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float temp = x[idx];
        x[idx] = y[idx];
        y[idx] = temp;
    }
}

void chatblas_sswap(int n, float *x, float *y) {
    float *d_x, *d_y;
    hipMalloc((void **)&d_x, n * sizeof(float));
    hipMalloc((void **)&d_y, n * sizeof(float));

    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    swap_kernel<<<numBlocks, blockSize>>>(n, d_x, d_y);

    hipMemcpy(x, d_x, n * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(y, d_y, n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
}
