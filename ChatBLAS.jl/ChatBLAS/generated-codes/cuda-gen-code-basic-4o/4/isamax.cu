#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void findMaxAbsKernel(int n, float *x, int *maxIndex, float *maxValue) {
    extern __shared__ float sharedData[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    sharedData[tid] = (i < n) ? fabsf(x[i]) : 0.0f;
    sharedData[blockDim.x + tid] = i;

    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s && sharedData[tid] < sharedData[tid + s]) {
            sharedData[tid] = sharedData[tid + s];
            sharedData[blockDim.x + tid] = sharedData[blockDim.x + tid + s];
        }
        __syncthreads();
    }
    
    if (tid == 0) {
        if (sharedData[0] > *maxValue) {
            *maxValue = sharedData[0];
            *maxIndex = (int)sharedData[blockDim.x];
        }
    }
}

int chatblas_isamax(int n, float *x) {
    if (n <= 0) return -1;

    int maxIndex = -1;
    float maxValue = 0.0f;
    float *d_x;
    int *d_maxIndex;
    float *d_maxValue;

    hipMalloc((void**)&d_x, n * sizeof(float));
    hipMalloc((void**)&d_maxIndex, sizeof(int));
    hipMalloc((void**)&d_maxValue, sizeof(float));
    
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_maxValue, &maxValue, sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;

    findMaxAbsKernel<<<gridSize, blockSize, 2 * blockSize * sizeof(float)>>>(n, d_x, d_maxIndex, d_maxValue);

    hipMemcpy(&maxIndex, d_maxIndex, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_maxIndex);
    hipFree(d_maxValue);

    return maxIndex;
}
