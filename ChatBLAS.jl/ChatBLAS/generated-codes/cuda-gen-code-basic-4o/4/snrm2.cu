#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void computeSquaredNorm(int n, float *x, float *result) {
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + tid;

    // Load elements into shared memory
    sdata[tid] = (i < n) ? x[i] * x[i] : 0.0f;
    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write the result for this block to global memory
    if (tid == 0) {
        atomicAdd(result, sdata[0]);
    }
}

float chatblas_snrm2(int n, float *x) {
    float *d_x, *d_result;
    float h_result = 0.0f;

    hipMalloc((void**)&d_x, n * sizeof(float));
    hipMalloc((void**)&d_result, sizeof(float));

    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_result, &h_result, sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    computeSquaredNorm<<<numBlocks, blockSize, blockSize * sizeof(float)>>>(n, d_x, d_result);

    hipMemcpy(&h_result, d_result, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_result);

    return sqrtf(h_result);
}
