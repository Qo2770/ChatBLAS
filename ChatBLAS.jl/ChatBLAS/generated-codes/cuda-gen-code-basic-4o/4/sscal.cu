#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#define THREADS_PER_BLOCK 256

// CUDA kernel to scale a vector
__global__ void sscal_kernel(int n, float a, float *x) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        x[idx] *= a;
    }
}

// Host function to configure and launch the CUDA kernel
void chatblas_sscal(int n, float a, float *x) {
    float *d_x;
    size_t size = n * sizeof(float);

    // Allocate memory on the GPU
    hipMalloc((void **)&d_x, size);

    // Copy the vector x to the GPU
    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);

    // Calculate the number of blocks needed
    int blocks = (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    // Launch the kernel
    sscal_kernel<<<blocks, THREADS_PER_BLOCK>>>(n, a, d_x);

    // Check for any errors arising while launching
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch sscal_kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Wait for the GPU to finish
    hipDeviceSynchronize();

    // Copy the result back to the host
    hipMemcpy(x, d_x, size, hipMemcpyDeviceToHost);

    // Free memory on the GPU
    hipFree(d_x);
}
