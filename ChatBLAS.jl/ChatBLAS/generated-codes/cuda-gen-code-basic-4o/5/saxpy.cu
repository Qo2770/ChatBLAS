#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void saxpyKernel(int n, float a, float *x, float *y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        y[index] = a * x[index] + y[index];
    }
}

void chatblas_saxpy(int n, float a, float *x, float *y) {
    float *d_x, *d_y;
    size_t size = n * sizeof(float);
    
    hipMalloc(&d_x, size);
    hipMalloc(&d_y, size);
    
    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);
    
    int blockSize = 256; // Number of threads in each block
    int numBlocks = (n + blockSize - 1) / blockSize; // Number of blocks
    
    saxpyKernel<<<numBlocks, blockSize>>>(n, a, d_x, d_y);
    
    hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);
    
    hipFree(d_x);
    hipFree(d_y);
}
