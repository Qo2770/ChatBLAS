#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void dotProductKernel(float *x, float *y, float *result, int n) {
    __shared__ float cache[256];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    
    float temp = 0.0;
    while (tid < n) {
        temp += x[tid] * y[tid];
        tid += blockDim.x * gridDim.x;
    }
    
    cache[cacheIndex] = temp;
    
    __syncthreads();
    
    int i = blockDim.x / 2;
    while (i != 0) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
        i /= 2;
    }
    
    if (cacheIndex == 0) {
        atomicAdd(result, cache[0]);
    }
}

float chatblas_sdot(int n, float *x, float *y) {
    float *d_x, *d_y, *d_result;
    float h_result = 0.0;
    
    hipMalloc((void**)&d_x, n * sizeof(float));
    hipMalloc((void**)&d_y, n * sizeof(float));
    hipMalloc((void**)&d_result, sizeof(float));
    
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_result, &h_result, sizeof(float), hipMemcpyHostToDevice);
    
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    
    dotProductKernel<<<blocksPerGrid, threadsPerBlock>>>(d_x, d_y, d_result, n);
    
    hipMemcpy(&h_result, d_result, sizeof(float), hipMemcpyDeviceToHost);
    
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_result);
    
    return h_result;
}
