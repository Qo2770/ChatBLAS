#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void compute_partial_norm(int n, float *x, float *partial_norm) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < n) {
        atomicAdd(partial_norm, x[idx] * x[idx]);
    }
}

float chatblas_snrm2(int n, float *x) {
    if (n <= 0) return 0.0f;

    // Allocate device memory
    float *d_x, *d_partial_norm;
    float h_partial_norm = 0.0f;

    hipMalloc((void**)&d_x, n * sizeof(float));
    hipMalloc((void**)&d_partial_norm, sizeof(float));

    // Copy data to device
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_partial_norm, &h_partial_norm, sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel to compute partial norms
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    compute_partial_norm<<<numBlocks, blockSize>>>(n, d_x, d_partial_norm);

    // Copy result back to host
    hipMemcpy(&h_partial_norm, d_partial_norm, sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_x);
    hipFree(d_partial_norm);

    // Compute square root of accumulated norm
    return sqrt(h_partial_norm);
}
