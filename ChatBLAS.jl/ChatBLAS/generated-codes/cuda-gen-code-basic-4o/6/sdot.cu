#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void dotProductKernel(float *x, float *y, float *result, int n) {
    __shared__ float cache[256]; // Assuming a maximum of 256 threads per block
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0.0f;
    while (tid < n) {
        temp += x[tid] * y[tid];
        tid += blockDim.x * gridDim.x;
    }

    // Store temp result in cache
    cache[cacheIndex] = temp;
    __syncthreads();

    // Reduce within block
    int i = blockDim.x / 2;
    while (i != 0) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
        i /= 2;
    }

    // Store block's result in global memory
    if (cacheIndex == 0) {
        atomicAdd(result, cache[0]);
    }
}

float chatblas_sdot(int n, float *x, float *y) {
    float *d_x, *d_y, *d_result;
    float h_result = 0.0f;

    hipMalloc((void **)&d_x, n * sizeof(float));
    hipMalloc((void **)&d_y, n * sizeof(float));
    hipMalloc((void **)&d_result, sizeof(float));

    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_result, &h_result, sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;

    dotProductKernel<<<gridSize, blockSize>>>(d_x, d_y, d_result, n);

    hipMemcpy(&h_result, d_result, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_result);

    return h_result;
}
