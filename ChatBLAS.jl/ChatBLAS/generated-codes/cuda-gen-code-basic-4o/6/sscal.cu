#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

// CUDA kernel to scale the vector
__global__ void scaleVectorKernel(int n, float a, float *x) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        x[index] = a * x[index];
    }
}

// Function to scale the vector using CUDA
void chatblas_sscal(int n, float a, float *x) {
    float *d_x;
    size_t size = n * sizeof(float);

    // Allocate memory on the device
    hipMalloc(&d_x, size);

    // Copy data from host to device
    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);

    // Define block and grid sizes
    int blockSize = 256; // Number of threads per block
    int numBlocks = (n + blockSize - 1) / blockSize; // Number of blocks needed

    // Launch the kernel
    scaleVectorKernel<<<numBlocks, blockSize>>>(n, a, d_x);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(x, d_x, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_x);
}
