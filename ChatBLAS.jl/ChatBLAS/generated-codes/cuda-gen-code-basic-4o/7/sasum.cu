#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sasum_kernel(int n, float *x, float *result) {
    extern __shared__ float sdata[];
    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + tid;
    float sum = 0.0f;

    // Accumulate absolute values in shared memory
    if (index < n) {
        sum = fabsf(x[index]);
    }
    sdata[tid] = sum;
    __syncthreads();

    // Reduce within block
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write block's result to global memory
    if (tid == 0) {
        atomicAdd(result, sdata[0]);
    }
}

float chatblas_sasum(int n, float *x) {
    float *d_x, *d_result;
    float h_result = 0.0f;

    // Allocate memory on the device
    hipMalloc((void**)&d_x, n * sizeof(float));
    hipMalloc((void**)&d_result, sizeof(float));
    
    // Copy data and initialize result on the device
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_result, &h_result, sizeof(float), hipMemcpyHostToDevice);

    // Kernel launch parameters
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;

    // Launch kernel
    sasum_kernel<<<gridSize, blockSize, blockSize * sizeof(float)>>>(n, d_x, d_result);

    // Copy the result back to the host
    hipMemcpy(&h_result, d_result, sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_x);
    hipFree(d_result);

    return h_result;
}
