#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA kernel that performs the copy operation
__global__ void vectorCopyKernel(int n, const float *x, float *y) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        y[idx] = x[idx];
    }
}

// Function that copies vector x to vector y
void chatblas_scopy(int n, float *x, float *y) {
    float *d_x, *d_y;
    size_t size = n * sizeof(float);

    // Allocate device memory
    hipMalloc((void **)&d_x, size);
    hipMalloc((void **)&d_y, size);

    // Copy x from host to device
    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);

    // Calculate block and grid sizes
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel
    vectorCopyKernel<<<blocksPerGrid, threadsPerBlock>>>(n, d_x, d_y);

    // Wait for the GPU to finish
    hipDeviceSynchronize();

    // Copy the result from device to host
    hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_x);
    hipFree(d_y);
}
