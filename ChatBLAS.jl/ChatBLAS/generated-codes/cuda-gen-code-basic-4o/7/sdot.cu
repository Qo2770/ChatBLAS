#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void dotProductKernel(int n, float *x, float *y, float *partialResults) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        partialResults[idx] = x[idx] * y[idx];
    }
}

float chatblas_sdot(int n, float *x, float *y) {
    float *d_x, *d_y, *d_partialResults;
    float *h_partialResults = (float *)malloc(n * sizeof(float));
    float result = 0.0f;

    hipMalloc((void **)&d_x, n * sizeof(float));
    hipMalloc((void **)&d_y, n * sizeof(float));
    hipMalloc((void **)&d_partialResults, n * sizeof(float));

    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;
    dotProductKernel<<<gridSize, blockSize>>>(n, d_x, d_y, d_partialResults);

    hipMemcpy(h_partialResults, d_partialResults, n * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++) {
        result += h_partialResults[i];
    }

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_partialResults);
    free(h_partialResults);

    return result;
}
