#include "hip/hip_runtime.h"
// chatblas_cuda.h
#ifndef CHATBLAS_CUDA_H
#define CHATBLAS_CUDA_H

#include <hip/hip_runtime.h>

// Add any additional declarations or helper functions that might be needed

#endif // CHATBLAS_CUDA_H
#include "chatblas_cuda.h"

// CUDA kernel to perform vector copy
__global__ void vectorCopyKernel(int n, const float *x, float *y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        y[index] = x[index];
    }
}

void chatblas_scopy(int n, float *x, float *y) {
    // Allocate GPU memory for input and output vectors
    float *d_x, *d_y;
    size_t size = n * sizeof(float);
    
    hipMalloc((void**)&d_x, size);
    hipMalloc((void**)&d_y, size);
    
    // Copy input vector from host to device
    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    
    // Define block and grid sizes
    int blockSize = 256; // You can set this to an optimal size based on your GPU architecture
    int gridSize = (n + blockSize - 1) / blockSize;
    
    // Launch kernel to perform vector copy on the GPU
    vectorCopyKernel<<<gridSize, blockSize>>>(n, d_x, d_y);
    
    // Copy result from device back to host
    hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);
    
    // Free device memory
    hipFree(d_x);
    hipFree(d_y);
}
