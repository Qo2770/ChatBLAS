#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void dotProductKernel(float *x, float *y, float *result, int n) {
    extern __shared__ float cache[];

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0.0f;
    while (tid < n) {
        temp += x[tid] * y[tid];
        tid += blockDim.x * gridDim.x;
    }

    cache[cacheIndex] = temp;
    __syncthreads();

    int i = blockDim.x / 2;
    while (i != 0) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0) {
        atomicAdd(result, cache[0]);
    }
}

float chatblas_sdot(int n, float *x, float *y) {
    float *d_x, *d_y, *d_result;
    float h_result = 0.0f;

    hipMalloc((void **)&d_x, n * sizeof(float));
    hipMalloc((void **)&d_y, n * sizeof(float));
    hipMalloc((void **)&d_result, sizeof(float));

    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_result, &h_result, sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    dotProductKernel<<<numBlocks, blockSize, blockSize * sizeof(float)>>>(d_x, d_y, d_result, n);

    hipMemcpy(&h_result, d_result, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_result);

    return h_result;
}
