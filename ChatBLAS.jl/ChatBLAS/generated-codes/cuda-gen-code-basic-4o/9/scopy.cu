#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

// CUDA kernel to copy elements from vector x to vector y
__global__ void vectorCopyKernel(int n, float *x, float *y) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        y[idx] = x[idx];
    }
}

void chatblas_scopy(int n, float *x, float *y) {
    // Check if there is work to do
    if (n <= 0) return;

    // Define CUDA kernel launch parameters
    int threadBlockSize = 256; // common choice for block size
    int numBlocks = (n + threadBlockSize - 1) / threadBlockSize;

    // Allocate device memory pointers
    float *d_x, *d_y;
    
    // Allocate device memory
    hipError_t err1 = hipMalloc((void**)&d_x, n * sizeof(float));
    hipError_t err2 = hipMalloc((void**)&d_y, n * sizeof(float));
    
    if (err1 != hipSuccess || err2 != hipSuccess) {
        fprintf(stderr, "CUDA memory allocation failed\n");
        // Handle error here (e.g., exit, or return an error code)
        return;
    }

    // Copy data from host to device
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);

    // Launch the kernel
    vectorCopyKernel<<<numBlocks, threadBlockSize>>>(n, d_x, d_y);

    // Ensure kernel launch was successful
    hipError_t errSync = hipGetLastError();
    hipError_t errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess) fprintf(stderr, "Kernel launch error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess) fprintf(stderr, "Kernel execution error: %s\n", hipGetErrorString(errAsync));

    // Copy result back from device to host
    hipMemcpy(y, d_y, n * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_x);
    hipFree(d_y);
}
