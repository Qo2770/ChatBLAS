#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void swap_vectors_kernel(int n, float *x, float *y) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float temp = x[idx];
        x[idx] = y[idx];
        y[idx] = temp;
    }
}

void chatblas_sswap(int n, float *x, float *y) {
    // Define the number of threads per block
    int threadsPerBlock = 256;
    // Calculate the number of blocks needed
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    // Allocate device memory
    float *d_x, *d_y;
    hipMalloc((void**)&d_x, n * sizeof(float));
    hipMalloc((void**)&d_y, n * sizeof(float));

    // Copy the input vectors from host to device
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);

    // Launch the kernel to swap the vectors
    swap_vectors_kernel<<<blocksPerGrid, threadsPerBlock>>>(n, d_x, d_y);

    // Copy the swapped vectors from device back to host
    hipMemcpy(x, d_x, n * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(y, d_y, n * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_x);
    hipFree(d_y);
}
