#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sasum_kernel(int n, float *x, float *sum) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  float partialSum = 0.0f;
  for (int i = tid; i < n; i += stride) {
    partialSum += abs(x[i]);
  }

  atomicAdd(sum, partialSum);
}

float chatblas_sasum(int n, float *x) {
  float *dev_x, *dev_sum;
  float sum = 0.0f;

  hipMalloc((void**)&dev_x, n * sizeof(float));
  hipMalloc((void**)&dev_sum, sizeof(float));

  hipMemcpy(dev_x, x, n * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dev_sum, &sum, sizeof(float), hipMemcpyHostToDevice);

  int blockSize = 256;
  int numBlocks = (n + blockSize - 1) / blockSize;

  sasum_kernel<<<numBlocks, blockSize>>>(n, dev_x, dev_sum);

  hipMemcpy(&sum, dev_sum, sizeof(float), hipMemcpyDeviceToHost);

  hipFree(dev_x);
  hipFree(dev_sum);

  return sum;
}