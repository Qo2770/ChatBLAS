#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void scopy_kernel(int n, float *x, float *y) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (tid < n) {
        y[tid] = x[tid];
    }
}

void chatblas_scopy(int n, float *x, float *y) {
    // Allocate GPU memory
    float *d_x, *d_y;
    hipMalloc((void **)&d_x, n * sizeof(float));
    hipMalloc((void **)&d_y, n * sizeof(float));
    
    // Copy input vectors from CPU to GPU
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    
    // Define block size and number of blocks
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    
    // Call the kernel
    scopy_kernel<<<numBlocks, blockSize>>>(n, d_x, d_y);
    
    // Copy the result vector from GPU to CPU
    hipMemcpy(y, d_y, n * sizeof(float), hipMemcpyDeviceToHost);
    
    // Free GPU memory
    hipFree(d_x);
    hipFree(d_y);
}
