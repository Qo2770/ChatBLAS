#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sscal_kernel(int n, float a, float *x) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        x[i] *= a;
    }
}

void chatblas_sscal(int n, float a, float *x) {
    float *d_x;
    int size = n * sizeof(float);

    hipMalloc((void **)&d_x, size);
    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    sscal_kernel<<<blocksPerGrid, threadsPerBlock>>>(n, a, d_x);

    hipMemcpy(x, d_x, size, hipMemcpyDeviceToHost);
    hipFree(d_x);
}
