#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sswap_kernel(int n, float *x, float *y) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < n) {
    float temp = x[i];
    x[i] = y[i];
    y[i] = temp;
  }
}

void chatblas_sswap(int n, float *x, float *y) {
  float *d_x, *d_y;

  // Allocate GPU memory
  hipMalloc(&d_x, n * sizeof(float));
  hipMalloc(&d_y, n * sizeof(float));

  // Transfer data from CPU to GPU
  hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);

  // Define the number of blocks and threads per block
  int threadsPerBlock = 256;
  int numBlocks = (n + threadsPerBlock - 1) / threadsPerBlock;

  // Call the kernel function
  sswap_kernel<<<numBlocks, threadsPerBlock>>>(n, d_x, d_y);

  // Transfer data from GPU to CPU
  hipMemcpy(x, d_x, n * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(y, d_y, n * sizeof(float), hipMemcpyDeviceToHost);

  // Free GPU memory
  hipFree(d_x);
  hipFree(d_y);
}
