#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void saxpy_kernel(int n, float a, float *x, float *y) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n)
        y[idx] = a * x[idx] + y[idx];
}

void chatblas_saxpy(int n, float a, float *x, float *y) {
    float *d_x, *d_y;
    size_t size = n * sizeof(float);

    hipMalloc((void **)&d_x, size);
    hipMalloc((void **)&d_y, size);

    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    saxpy_kernel<<<blocksPerGrid, threadsPerBlock>>>(n, a, d_x, d_y);

    hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
}