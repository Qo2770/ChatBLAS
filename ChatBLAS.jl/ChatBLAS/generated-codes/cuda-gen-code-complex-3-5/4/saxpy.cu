#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void saxpy_kernel(int n, float a, float *x, float *y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride) {
        y[i] += a * x[i];
    }
}

void chatblas_saxpy(int n, float a, float *x, float *y) {
    float *d_x, *d_y;
    size_t size = n * sizeof(float);

    hipMalloc((void **)&d_x, size);
    hipMalloc((void **)&d_y, size);

    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    saxpy_kernel<<<blocksPerGrid, threadsPerBlock>>>(n, a, d_x, d_y);

    hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
}
