#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sdot_kernel(int n, float *x, float *y, float *res) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ float shared_res[256];

    if (tid < n) {
        shared_res[threadIdx.x] = x[tid] * y[tid];
    } else {
        shared_res[threadIdx.x] = 0.0f;
    }

    __syncthreads();

    int i = blockDim.x / 2;
    while (i != 0) {
        if (threadIdx.x < i) {
            shared_res[threadIdx.x] += shared_res[threadIdx.x + i];
        }
        __syncthreads();
        i /= 2;
    }

    if (threadIdx.x == 0) {
        atomicAdd(res, shared_res[0]);
    }
}

float chatblas_sdot(int n, float *x, float *y) {
    float *d_x, *d_y, *d_res;
    float res = 0.0f;

    hipMalloc((void **)&d_x, n * sizeof(float));
    hipMalloc((void **)&d_y, n * sizeof(float));
    hipMalloc((void **)&d_res, sizeof(float));

    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    sdot_kernel<<<blocksPerGrid, threadsPerBlock>>>(n, d_x, d_y, d_res);

    hipMemcpy(&res, d_res, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_res);

    return res;
}
