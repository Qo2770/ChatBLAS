#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void saxpy_kernel(int n, float a, float *x, float *y) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < n) {
    y[tid] = a * x[tid] + y[tid];
  }
}

void chatblas_saxpy(int n, float a, float *x, float *y) {
  float *device_x, *device_y;
  hipMalloc((void**)&device_x, sizeof(float) * n);
  hipMalloc((void**)&device_y, sizeof(float) * n);

  hipMemcpy(device_x, x, sizeof(float) * n, hipMemcpyHostToDevice);
  hipMemcpy(device_y, y, sizeof(float) * n, hipMemcpyHostToDevice);

  int blockSize = 256;
  int numBlocks = (n + blockSize - 1) / blockSize;

  saxpy_kernel<<<numBlocks, blockSize>>>(n, a, device_x, device_y);

  hipMemcpy(y, device_y, sizeof(float) * n, hipMemcpyDeviceToHost);

  hipFree(device_x);
  hipFree(device_y);
}
