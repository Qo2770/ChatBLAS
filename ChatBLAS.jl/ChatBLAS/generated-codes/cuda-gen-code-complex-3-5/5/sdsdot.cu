#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sdsdot_kernel(int n, float b, float *x, float *y, float *res) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    double sum = 0.0;
    for (int i = tid; i < n; i += stride) {
        sum += (double)x[i] * (double)y[i];
    }

    atomicAdd(res, (float)sum);
}

float chatblas_sdsdot(int n, float b, float *x, float *y) {
    float *d_x, *d_y, *d_res;
    float h_res = 0.0;

    hipMalloc((void**)&d_x, sizeof(float) * n);
    hipMalloc((void**)&d_y, sizeof(float) * n);
    hipMalloc((void**)&d_res, sizeof(float));

    hipMemcpy(d_x, x, sizeof(float) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, sizeof(float) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_res, &h_res, sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    sdsdot_kernel<<<numBlocks, blockSize>>>(n, b, d_x, d_y, d_res);

    hipMemcpy(&h_res, d_res, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_res);

    return h_res + b;
}
