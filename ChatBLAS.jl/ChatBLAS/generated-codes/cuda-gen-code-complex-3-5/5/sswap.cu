#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sswap_kernel(int n, float *x, float *y) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < n) {
    float temp = x[tid];
    x[tid] = y[tid];
    y[tid] = temp;
  }
}

void chatblas_sswap(int n, float *x, float *y) {
  int numThreadsPerBlock = 256;
  int numBlocks = (n + numThreadsPerBlock - 1) / numThreadsPerBlock;
  
  float *d_x, *d_y;
  hipMalloc((void **)&d_x, n * sizeof(float));
  hipMalloc((void **)&d_y, n * sizeof(float));
  
  hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);
  
  sswap_kernel<<<numBlocks, numThreadsPerBlock>>>(n, d_x, d_y);
  
  hipMemcpy(x, d_x, n * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(y, d_y, n * sizeof(float), hipMemcpyDeviceToHost);
  
  hipFree(d_x);
  hipFree(d_y);
}
