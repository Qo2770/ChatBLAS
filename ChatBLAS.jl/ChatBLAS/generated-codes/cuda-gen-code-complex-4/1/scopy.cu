#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void scopy_kernel( int n, float *x, float *y ) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if(i < n){
      y[i] = x[i];
   }
}

void chatblas_scopy(int n, float *x, float *y) {
   int size = n*sizeof(float);
   float *d_x, *d_y;
   hipMalloc((void**)&d_x, size);
   hipMalloc((void**)&d_y, size);
   hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
   int blockSize = 256;
   int numBlocks = (n + blockSize - 1) / blockSize;
   scopy_kernel<<<numBlocks, blockSize>>>(n, d_x, d_y);
   hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);
   hipFree(d_x);
   hipFree(d_y);
}