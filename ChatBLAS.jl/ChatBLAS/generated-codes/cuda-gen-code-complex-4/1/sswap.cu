#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sswap_kernel(int n, float *x, float *y) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        float a = x[idx];
        x[idx] = y[idx];
        y[idx] = a;
    }
}

void chatblas_sswap(int n, float *x, float *y) {
    float *d_x, *d_y;
    int size = n * sizeof(float);
    hipMalloc((void **) &d_x, size);
    hipMalloc((void **) &d_y, size);
    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid =(n + threadsPerBlock - 1) / threadsPerBlock;
    sswap_kernel<<<blocksPerGrid, threadsPerBlock>>>(n, d_x, d_y);
    
    hipMemcpy(x, d_x, size, hipMemcpyDeviceToHost);
    hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);
    hipFree(d_x);
    hipFree(d_y);
}