#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sscal_kernel( int n, float a , float *x ) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        x[i] = a * x[i];
    }
}

void chatblas_sscal( int n, float a, float *x) {
    float *d_x;

    hipMalloc(&d_x, n * sizeof(float));
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    sscal_kernel<<<numBlocks, blockSize>>>(n, a, d_x);

    hipMemcpy(x, d_x, n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
}