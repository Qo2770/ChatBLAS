#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sasum_kernel(int n, float *x, float *sum) {

int i = blockDim.x * blockIdx.x + threadIdx.x;

if (i < n) {
atomicAdd(sum, fabs(x[i]));
}
}

float chatblas_sasum(int n, float *x) {
float *d_x, *d_sum;
float *h_sum = (float*)malloc(sizeof(float));

hipMalloc((void **)&d_x, sizeof(float)*n);
hipMalloc((void **)&d_sum, sizeof(float));

hipMemcpy(d_x, x, sizeof(float)*n, hipMemcpyHostToDevice);

dim3 DimBlock(256, 1, 1);
dim3 DimGrid((n-1)/256 + 1, 1, 1);

sasum_kernel<<<DimGrid, DimBlock>>>(n, d_x, d_sum);
hipDeviceSynchronize();

hipMemcpy(h_sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);

float sum = *h_sum;
free(h_sum);
hipFree(d_x); hipFree(d_sum);

return sum;
}