#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void saxpy_kernel(int n, float a, float *x, float *y) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) y[i] = a * x[i] + y[i];
}

void chatblas_saxpy(int n, float a, float *x, float *y) {
  int size = n * sizeof(float);
  float *d_x, *d_y;

  hipMalloc(&d_x, size);
  hipMalloc(&d_y, size);

  hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

  saxpy_kernel<<<(n+255)/256, 256>>>(n, a, d_x, d_y);

  hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);

  hipFree(d_x);
  hipFree(d_y);
}