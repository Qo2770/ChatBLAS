#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void scopy_kernel( int n, float *x, float *y ) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i < n ) y[i] = x[i];
}

void chatblas_scopy(int n, float *x, float *y) {
    float* x_d;
    float* y_d;

    hipMalloc((void **)&x_d, n*sizeof(float));
    hipMalloc((void **)&y_d, n*sizeof(float));
  
    hipMemcpy(x_d, x, n*sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    scopy_kernel<<<numBlocks, blockSize>>>(n, x_d, y_d);
  
    hipMemcpy(y, y_d, n*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(x_d);
    hipFree(y_d);
}