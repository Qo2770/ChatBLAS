#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sscal_kernel( int n, float a , float *x ) {
    int indx = threadIdx.x + blockIdx.x * blockDim.x;
    if (indx < n) 
        x[indx] *= a;
}

void chatblas_sscal( int n, float a, float *x) {

    float *d_x;
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    hipMalloc((void**)&d_x, n * sizeof(float));
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);

    sscal_kernel<<<numBlocks, blockSize>>>(n, a, d_x);

    hipMemcpy(x, d_x, n *sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_x);
}