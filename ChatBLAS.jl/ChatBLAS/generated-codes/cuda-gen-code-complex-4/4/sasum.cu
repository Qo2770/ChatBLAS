#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sasum_kernel(int n, float *x, float *sum)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    __shared__ float cache[256];

    float temp = 0;
    while (index < n) {
        temp += fabs(x[index]);
        index += stride;
    }

    cache[threadIdx.x] = temp;
    __syncthreads();

    if (threadIdx.x == 0) {
        float temp = 0;
        for (int i = 0; i < blockDim.x; i++)
            temp += cache[i];
        atomicAdd(sum, temp);
    }
}

float chatblas_sasum(int n, float *x)
{
    float *x_device, *sum_device;
    float sum_host = 0;

    hipMalloc((void**)&x_device, n * sizeof(float));
    hipMemcpy(x_device, x, n * sizeof(float), hipMemcpyHostToDevice);
    
    hipMalloc((void**)&sum_device, sizeof(float));
    hipMemcpy(sum_device, &sum_host, sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    sasum_kernel<<<numBlocks, blockSize>>>(n, x_device, sum_device);

    hipMemcpy(&sum_host, sum_device, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(x_device);
    hipFree(sum_device);

    return sum_host;
}