#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sdsdot_kernel( int n, float b, float *x, float *y, float *res ) { 
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  double temp_sum = 0.0;
  if (i < n) {
    temp_sum += ((double)x[i]) * ((double)y[i]);
  }
  atomicAdd(res, (float)temp_sum);
} 

float chatblas_sdsdot( int n, float b, float *x, float *y) {
  float *x_gpu, *y_gpu, *res_gpu; 
  float res = b;

  int blockSize = 256;
  int gridSize = (int)ceil((float)n/blockSize);

  hipMalloc((void**)&x_gpu, n * sizeof(float));
  hipMalloc((void**)&y_gpu, n * sizeof(float));
  hipMalloc((void**)&res_gpu, sizeof(float));

  hipMemcpy(x_gpu, x, n * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(y_gpu, y, n * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(res_gpu, &res, sizeof(float), hipMemcpyHostToDevice);

  sdsdot_kernel<<<gridSize, blockSize>>>(n, b, x_gpu, y_gpu, res_gpu);
  
  hipMemcpy(&res, res_gpu, sizeof(float), hipMemcpyDeviceToHost);

  hipFree(x_gpu); 
  hipFree(y_gpu); 
  hipFree(res_gpu);

  return res;
}