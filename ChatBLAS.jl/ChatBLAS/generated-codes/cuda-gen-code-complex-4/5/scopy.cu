#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void scopy_kernel( int n, float *x, float *y ) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        y[idx] = x[idx];
    }
}

void chatblas_scopy(int n, float *x, float *y) {
    float *d_x, *d_y;

    int size = n * sizeof(float);
    hipMalloc((void**)&d_x, size);
    hipMalloc((void**)&d_y, size);

    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);

    int blocks = (n + 255) / 256;
    scopy_kernel <<< blocks, 256 >>> (n, d_x, d_y);

    hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
}
