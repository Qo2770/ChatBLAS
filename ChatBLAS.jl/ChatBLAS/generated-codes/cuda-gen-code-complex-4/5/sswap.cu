#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sswap_kernel(int n, float *x, float *y) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i < n) {
        float tmp = x[i];
        x[i] = y[i];
        y[i] = tmp;
    }
}

void chatblas_sswap(int n, float *x, float *y) {
    float *dx, *dy;

    hipMalloc(&dx, n*sizeof(float));
    hipMemcpy(dx, x, n*sizeof(float), hipMemcpyHostToDevice);

    hipMalloc(&dy, n*sizeof(float));
    hipMemcpy(dy, y, n*sizeof(float), hipMemcpyHostToDevice);

    int blocks = (n + 255) / 256;
    int threads = (n < 256) ? n : 256;

    sswap_kernel<<<blocks, threads>>>(n, dx, dy);
        
    hipMemcpy(x, dx, n*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(y, dy, n*sizeof(float), hipMemcpyDeviceToHost);
        
    hipFree(dx);
    hipFree(dy);
}