#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sdot_kernel(int n, float *x, float *y, float *res) {
    extern __shared__ float shared_mem[];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int local_tid = threadIdx.x;

    shared_mem[local_tid] = (tid < n) ? x[tid] * y[tid] : 0.0f;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (local_tid < stride) {
            shared_mem[local_tid] += shared_mem[local_tid + stride];
        }
        __syncthreads();
    }

    if (local_tid == 0) {
        atomicAdd(res, shared_mem[0]);
    }
}

float chatblas_sdot(int n, float *x, float *y) {
    float *d_x, *d_y, *d_res;
    float h_res = 0.0f;

    hipMalloc((void**)&d_x, n * sizeof(float));
    hipMalloc((void**)&d_y, n * sizeof(float));
    hipMalloc((void**)&d_res, sizeof(float));

    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_res, &h_res, sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;
    sdot_kernel<<<gridSize, blockSize, blockSize * sizeof(float)>>>(n, d_x, d_y, d_res);

    hipMemcpy(&h_res, d_res, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_res);

    return h_res;
}