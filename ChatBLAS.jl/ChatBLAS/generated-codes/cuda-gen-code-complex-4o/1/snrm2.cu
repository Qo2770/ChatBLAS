#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void snrm2_kernel(int n, float *x, float *res) {
    extern __shared__ float cache[];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0.0;
    while (tid < n) {
        temp += x[tid] * x[tid];
        tid += blockDim.x * gridDim.x;
    }

    cache[cacheIndex] = temp;
    __syncthreads();

    int i = blockDim.x / 2;
    while (i != 0) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0) {
        atomicAdd(res, cache[0]);
    }
}

float chatblas_snrm2(int n, float *x) {
    float *d_x, *d_res;
    float result;
    
    hipMalloc((void**)&d_x, n * sizeof(float));
    hipMalloc((void**)&d_res, sizeof(float));
    
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(d_res, 0, sizeof(float));

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    snrm2_kernel<<<numBlocks, blockSize, blockSize * sizeof(float)>>>(n, d_x, d_res);

    hipMemcpy(&result, d_res, sizeof(float), hipMemcpyDeviceToHost);
    
    result = sqrt(result);

    hipFree(d_x);
    hipFree(d_res);

    return result;
}