#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sdsdot_kernel( int n, float b, float *x, float *y, float *res ) {
    extern __shared__ double sdata[];
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int s = blockDim.x / 2; 
    double sum = 0.0;
    
    while (tid < n) {
        sum += (double)x[tid] * (double)y[tid];
        tid += blockDim.x * gridDim.x;
    }

    sdata[threadIdx.x] = sum;
    __syncthreads();

    for (int stride = s; stride > 0; stride /= 2) { 
        if (threadIdx.x < stride) 
            sdata[threadIdx.x] += sdata[threadIdx.x + stride];
        __syncthreads();
    }
    
    if (threadIdx.x == 0)
        atomicAdd(res, sdata[0]);
}

float chatblas_sdsdot( int n, float b, float *x, float *y) {
    float *d_x, *d_y, *d_res;
    float result = 0.0;
    
    hipMalloc((void**)&d_x, n * sizeof(float));
    hipMalloc((void**)&d_y, n * sizeof(float));
    hipMalloc((void**)&d_res, sizeof(float));
    
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_res, &result, sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;
    
    sdsdot_kernel<<<gridSize, blockSize, blockSize * sizeof(double)>>>(n, b, d_x, d_y, d_res);
    
    hipMemcpy(&result, d_res, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_res);
    
    return result + b;
}