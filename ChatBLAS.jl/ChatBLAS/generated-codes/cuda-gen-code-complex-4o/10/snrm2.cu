#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void snrm2_kernel(int n, float *x, float *res) {
    __shared__ float cache[256];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    float temp = 0.0;
    while (tid < n) {
        temp += x[tid] * x[tid];
        tid += blockDim.x * gridDim.x;
    }
    cache[cacheIndex] = temp;
    __syncthreads();
    int i = blockDim.x / 2;
    while (i != 0) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
        i /= 2;
    }
    if (cacheIndex == 0) {
        atomicAdd(res, cache[0]);
    }
}

float chatblas_snrm2(int n, float *x) {
    float *d_x, *d_res;
    float h_res = 0.0;
    hipMalloc((void**)&d_x, n * sizeof(float));
    hipMalloc((void**)&d_res, sizeof(float));
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_res, &h_res, sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    snrm2_kernel<<<numBlocks, blockSize>>>(n, d_x, d_res);

    hipMemcpy(&h_res, d_res, sizeof(float), hipMemcpyDeviceToHost);
    
    hipFree(d_x);
    hipFree(d_res);
    
    return sqrt(h_res);
}