#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sscal_kernel(int n, float a, float *x) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        x[idx] *= a;
    }
}

void chatblas_sscal(int n, float a, float *x) {
    float *d_x;
    size_t size = n * sizeof(float);

    hipMalloc((void**)&d_x, size);
    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    sscal_kernel<<<blocksPerGrid, threadsPerBlock>>>(n, a, d_x);

    hipMemcpy(x, d_x, size, hipMemcpyDeviceToHost);
    hipFree(d_x);
}