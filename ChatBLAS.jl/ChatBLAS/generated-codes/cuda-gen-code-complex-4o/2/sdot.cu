#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"
__global__ void sdot_kernel(int n, float *x, float *y, float *res) {
    extern __shared__ float cache[];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    float temp = 0;

    while (tid < n) {
        temp += x[tid] * y[tid];
        tid += blockDim.x * gridDim.x;
    }

    cache[cacheIndex] = temp;
    __syncthreads();

    int i = blockDim.x / 2;
    while (i != 0) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0) {
        atomicAdd(res, cache[0]);
    }
}

float chatblas_sdot(int n, float *x, float *y) {
    float *d_x, *d_y, *d_res;
    float h_res = 0.0f;
    int size = n * sizeof(float);
    
    hipMalloc((void**)&d_x, size);
    hipMalloc((void**)&d_y, size);
    hipMalloc((void**)&d_res, sizeof(float));

    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);
    hipMemcpy(d_res, &h_res, sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    size_t sharedMemSize = blockSize * sizeof(float);

    sdot_kernel<<<numBlocks, blockSize, sharedMemSize>>>(n, d_x, d_y, d_res);
    hipMemcpy(&h_res, d_res, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_res);

    return h_res;
}