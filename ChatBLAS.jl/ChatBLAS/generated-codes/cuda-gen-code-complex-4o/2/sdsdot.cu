#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sdsdot_kernel(int n, float b, float *x, float *y, float *res) {
    extern __shared__ double sdata[];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int blockSize = blockDim.x;
    int tx = threadIdx.x;
    
    double temp = 0.0;
    while (tid < n) {
        temp += (double)x[tid] * (double)y[tid];
        tid += blockDim.x * gridDim.x;
    }

    sdata[tx] = temp;
    __syncthreads();

    for (unsigned int s = blockSize / 2; s > 0; s >>= 1) {
        if (tx < s) {
            sdata[tx] += sdata[tx + s];
        }
        __syncthreads();
    }

    if (tx == 0) {
        atomicAdd(res, sdata[0]);
    }
}

float chatblas_sdsdot(int n, float b, float *x, float *y) {
    float *d_x, *d_y, *d_res;
    float host_res = 0.0f;
    hipMalloc((void**)&d_x, n * sizeof(float));
    hipMalloc((void**)&d_y, n * sizeof(float));
    hipMalloc((void**)&d_res, sizeof(float));
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_res, &host_res, sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;
    sdsdot_kernel<<<gridSize, blockSize, blockSize * sizeof(double)>>>(n, b, d_x, d_y, d_res);
    
    hipMemcpy(&host_res, d_res, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_res);
    
    return host_res + b;
}