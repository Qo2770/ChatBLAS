#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sswap_kernel(int n, float *x, float *y) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float temp = x[idx];
        x[idx] = y[idx];
        y[idx] = temp;
    }
}

void chatblas_sswap(int n, float *x, float *y) {
    float *d_x, *d_y;
    size_t size = n * sizeof(float);
    
    hipMalloc((void**)&d_x, size);
    hipMalloc((void**)&d_y, size);
    
    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);
    
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    
    sswap_kernel<<<numBlocks, blockSize>>>(n, d_x, d_y);
    
    hipMemcpy(x, d_x, size, hipMemcpyDeviceToHost);
    hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);
    
    hipFree(d_x);
    hipFree(d_y);
}