#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sasum_kernel(int n, float *x, float *sum) {
    extern __shared__ float sdata[];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    float tmp = 0;

    while (i < n) {
        tmp += fabsf(x[i]);
        i += blockDim.x * gridDim.x;
    }
    
    sdata[tid] = tmp;
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        atomicAdd(sum, sdata[0]);
    }
}

float chatblas_sasum(int n, float *x) {
    float *d_x, *d_sum;
    float h_sum = 0;
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    hipMalloc((void **)&d_x, n * sizeof(float));
    hipMalloc((void **)&d_sum, sizeof(float));

    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(d_sum, 0, sizeof(float));

    sasum_kernel<<<numBlocks, blockSize, blockSize * sizeof(float)>>>(n, d_x, d_sum);
    hipMemcpy(&h_sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_sum);

    return h_sum;
}