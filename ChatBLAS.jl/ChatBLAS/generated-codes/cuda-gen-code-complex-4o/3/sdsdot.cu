#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sdsdot_kernel( int n, float b, float *x, float *y, float *res ) {
    extern __shared__ double sdata[];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;
    double temp = 0;

    if (i < n) {
        temp = (double)x[i] * (double)y[i];
    }
    sdata[tid] = temp;
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        atomicAdd(res, sdata[0]);
    }
}

float chatblas_sdsdot( int n, float b, float *x, float *y) {
    float *d_x, *d_y;
    float *d_res, h_res;
    hipMalloc((void**)&d_x, n * sizeof(float));
    hipMalloc((void**)&d_y, n * sizeof(float));
    hipMalloc((void**)&d_res, sizeof(float));

    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(d_res, 0, sizeof(float));

    int blockSize = 256; 
    int gridSize = (n + blockSize - 1) / blockSize;
    size_t sharedMemorySize = blockSize * sizeof(double);

    sdsdot_kernel<<<gridSize, blockSize, sharedMemorySize>>>(n, b, d_x, d_y, d_res);

    hipMemcpy(&h_res, d_res, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_res);

    return h_res + b;
}