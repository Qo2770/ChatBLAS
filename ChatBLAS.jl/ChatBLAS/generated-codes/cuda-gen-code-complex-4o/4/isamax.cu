#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void isamax_kernel(int n, float *x, int *ind) {
    extern __shared__ float sdata[];
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + tid;

    sdata[tid] = (idx < n) ? fabsf(x[idx]) : -1.0;
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s && sdata[tid] < sdata[tid + s]) {
            sdata[tid] = sdata[tid + s];
            ind[tid] = ind[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        ind[blockIdx.x] = idx - blockDim.x + threadIdx.x;
    }
}

int chatblas_isamax(int n, float *x) {
    float *d_x;
    int *d_ind, *h_ind;
    int max_blocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    h_ind = (int *)malloc(max_blocks * sizeof(int));

    hipMalloc((void **)&d_x, n * sizeof(float));
    hipMalloc((void **)&d_ind, max_blocks * sizeof(int));

    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);

    isamax_kernel<<<max_blocks, BLOCK_SIZE, BLOCK_SIZE * sizeof(float)>>>(n, d_x, d_ind);

    hipMemcpy(h_ind, d_ind, max_blocks * sizeof(int), hipMemcpyDeviceToHost);

    int max_ind = h_ind[0];
    for (int i = 1; i < max_blocks; i++) {
        if (fabs(x[h_ind[i]]) > fabs(x[max_ind])) {
            max_ind = h_ind[i];
        }
    }

    hipFree(d_x);
    hipFree(d_ind);
    free(h_ind);

    return max_ind;
}