#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sdot_kernel(int n, float *x, float *y, float *res) {
    extern __shared__ float sdata[];
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int index = tid;
    float sum = 0.0f;
    
    while (index < n) {
        sum += x[index] * y[index];
        index += blockDim.x * gridDim.x;
    }
    
    sdata[threadIdx.x] = sum;
    __syncthreads();
    
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            sdata[threadIdx.x] += sdata[threadIdx.x + s];
        }
        __syncthreads();
    }
    
    if (threadIdx.x == 0) {
        atomicAdd(res, sdata[0]);
    }
}

float chatblas_sdot(int n, float *x, float *y) {
    float *d_x, *d_y, *d_res;
    float h_res = 0.0f;
    hipMalloc((void **)&d_x, n * sizeof(float));
    hipMalloc((void **)&d_y, n * sizeof(float));
    hipMalloc((void **)&d_res, sizeof(float));
    
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_res, &h_res, sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;
    sdot_kernel<<<gridSize, blockSize, blockSize * sizeof(float)>>>(n, d_x, d_y, d_res);
    
    hipMemcpy(&h_res, d_res, sizeof(float), hipMemcpyDeviceToHost);
    
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_res);
    
    return h_res;
}