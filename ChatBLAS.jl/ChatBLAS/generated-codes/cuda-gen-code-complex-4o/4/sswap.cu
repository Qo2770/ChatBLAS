#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sswap_kernel(int n, float *x, float *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        float temp = x[i];
        x[i] = y[i];
        y[i] = temp;
    }
}

void chatblas_sswap(int n, float *x, float *y) {
    float *d_x, *d_y;
    size_t size = n * sizeof(float);

    hipMalloc((void**)&d_x, size);
    hipMalloc((void**)&d_y, size);

    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    sswap_kernel<<<numBlocks, blockSize>>>(n, d_x, d_y);

    hipMemcpy(x, d_x, size, hipMemcpyDeviceToHost);
    hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
}