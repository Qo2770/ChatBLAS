#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sasum_kernel(int n, float *x, float *sum) {
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        sdata[tid] = fabsf(x[i]);
    } else {
        sdata[tid] = 0;
    }
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        atomicAdd(sum, sdata[0]);
    }
}

float chatblas_sasum(int n, float *x) {
    float *d_x, *d_sum, h_sum = 0.0f;
    hipMalloc((void**)&d_x, n * sizeof(float));
    hipMalloc((void**)&d_sum, sizeof(float));
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_sum, &h_sum, sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    sasum_kernel<<<numBlocks, blockSize, blockSize * sizeof(float)>>>(n, d_x, d_sum);

    hipMemcpy(&h_sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_x);
    hipFree(d_sum);
    
    return h_sum;
}