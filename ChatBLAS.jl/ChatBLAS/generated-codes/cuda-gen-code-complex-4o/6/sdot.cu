#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sdot_kernel(int n, float *x, float *y, float *res) {
    extern __shared__ float cache[];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0;
    while (tid < n) {
        temp += x[tid] * y[tid];
        tid += blockDim.x * gridDim.x;
    }

    cache[cacheIndex] = temp;
    __syncthreads();

    int i = blockDim.x / 2;
    while (i != 0) {
        if (cacheIndex < i) cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0) atomicAdd(res, cache[0]);
}

float chatblas_sdot(int n, float *x, float *y) {
    float *d_x, *d_y, *d_res;
    float result = 0;
    hipMalloc((void **)&d_x, n * sizeof(float));
    hipMalloc((void **)&d_y, n * sizeof(float));
    hipMalloc((void **)&d_res, sizeof(float));

    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_res, &result, sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;
    sdot_kernel<<<gridSize, blockSize, blockSize * sizeof(float)>>>(n, d_x, d_y, d_res);

    hipMemcpy(&result, d_res, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_res);
    
    return result;
}