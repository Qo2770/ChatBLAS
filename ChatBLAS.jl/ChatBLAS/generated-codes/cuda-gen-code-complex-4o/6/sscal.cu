#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sscal_kernel(int n, float a, float *x) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        x[idx] *= a;
    }
}

void chatblas_sscal(int n, float a, float *x) {
    float *d_x;
    hipMalloc((void **)&d_x, n * sizeof(float));
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    sscal_kernel<<<numBlocks, blockSize>>>(n, a, d_x);

    hipMemcpy(x, d_x, n * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_x);
}