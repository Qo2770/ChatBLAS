#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sdsdot_kernel( int n, float b, float *x, float *y, float *res ) {
    __shared__ double cache[256];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    
    double temp = 0.0;
    while (tid < n) {
        temp += (double)x[tid] * (double)y[tid];
        tid += blockDim.x * gridDim.x;
    }

    cache[cacheIndex] = temp;

    __syncthreads();

    int i = blockDim.x / 2;
    while (i != 0) {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0) {
        atomicAdd(res, cache[0] + (double)b);
    }
}

float chatblas_sdsdot( int n, float b, float *x, float *y) {
    float *d_x, *d_y;
    float *d_res, h_res;

    hipMalloc((void**)&d_x, n * sizeof(float));
    hipMalloc((void**)&d_y, n * sizeof(float));
    hipMalloc((void**)&d_res, sizeof(float));

    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);

    float initial = 0.0f;
    hipMemcpy(d_res, &initial, sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    sdsdot_kernel<<<numBlocks, blockSize>>>(n, b, d_x, d_y, d_res);
    hipDeviceSynchronize();

    hipMemcpy(&h_res, d_res, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_res);

    return h_res;
}