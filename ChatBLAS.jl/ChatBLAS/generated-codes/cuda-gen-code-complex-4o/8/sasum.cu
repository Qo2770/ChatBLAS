#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"
__global__ void sasum_kernel(int n, float *x, float *sum) {
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = gridDim.x * blockDim.x;
    sdata[tid] = 0.0;

    while (index < n) {
        sdata[tid] += fabsf(x[index]);
        index += stride;
    }

    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        atomicAdd(sum, sdata[0]);
    }
}

float chatblas_sasum(int n, float *x) {
    float *d_x, *d_sum;
    float h_sum = 0.0;

    hipMalloc((void**)&d_x, n * sizeof(float));
    hipMalloc((void**)&d_sum, sizeof(float));

    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_sum, &h_sum, sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    sasum_kernel<<<blocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(float)>>>(n, d_x, d_sum);

    hipMemcpy(&h_sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_sum);

    return h_sum;
}