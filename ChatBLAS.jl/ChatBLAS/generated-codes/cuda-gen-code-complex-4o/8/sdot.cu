#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sdot_kernel(int n, float *x, float *y, float *res) {
    extern __shared__ float shmem[];

    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    float temp_sum = 0.0f;

    if (index < n) {
        temp_sum = x[index] * y[index];
    }

    shmem[tid] = temp_sum;
    __syncthreads();

    // Reduction in shared memory
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            shmem[tid] += shmem[tid + s];
        }
        __syncthreads();
    }

    // Store the result in the first element of the block's shared memory
    if (tid == 0) {
        atomicAdd(res, shmem[0]);
    }
}

float chatblas_sdot(int n, float *x, float *y) {
    float *d_x, *d_y, *d_res;
    float h_res = 0.0f;

    hipMalloc((void **)&d_x, n * sizeof(float));
    hipMalloc((void **)&d_y, n * sizeof(float));
    hipMalloc((void **)&d_res, sizeof(float));

    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_res, &h_res, sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    sdot_kernel<<<numBlocks, blockSize, blockSize * sizeof(float)>>>(n, d_x, d_y, d_res);

    hipMemcpy(&h_res, d_res, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_res);

    return h_res;
}