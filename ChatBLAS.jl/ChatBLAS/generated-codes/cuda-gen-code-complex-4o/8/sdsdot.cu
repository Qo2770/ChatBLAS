#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sdsdot_kernel( int n, float b, float *x, float *y, float *res ) {
    extern __shared__ double shared_mem[];
    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + tid;
    double sum = 0.0;
    
    for (int i = index; i < n; i += blockDim.x * gridDim.x) {
        sum += (double)x[i] * (double)y[i];
    }
    
    shared_mem[tid] = sum;
    __syncthreads();
    
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_mem[tid] += shared_mem[tid + stride];
        }
        __syncthreads();
    }
    
    if (tid == 0) {
        atomicAdd(res, shared_mem[0]);
    }
}

float chatblas_sdsdot( int n, float b, float *x, float *y) {
    float *d_x, *d_y, *d_res;
    float h_res = 0.0f;
    hipMalloc((void**)&d_x, n * sizeof(float));
    hipMalloc((void**)&d_y, n * sizeof(float));
    hipMalloc((void**)&d_res, sizeof(float));
    
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_res, &h_res, sizeof(float), hipMemcpyHostToDevice);
    
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    sdsdot_kernel<<<blocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(double)>>>(n, b, d_x, d_y, d_res);
    
    hipMemcpy(&h_res, d_res, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_res);
    
    return h_res + b;
}