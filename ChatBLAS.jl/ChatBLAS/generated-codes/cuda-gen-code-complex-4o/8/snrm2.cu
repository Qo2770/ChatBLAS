#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void snrm2_kernel(int n, float *x, float *res) {
    __shared__ float cache[256];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    float temp = 0.0;

    while (tid < n) {
        temp += x[tid] * x[tid];
        tid += blockDim.x * gridDim.x;
    }

    cache[threadIdx.x] = temp;

    __syncthreads();

    int i = blockDim.x / 2;
    while (i != 0) {
        if (threadIdx.x < i) {
            cache[threadIdx.x] += cache[threadIdx.x + i];
        }
        __syncthreads();
        i /= 2;
    }

    if (threadIdx.x == 0) {
        atomicAdd(res, cache[0]);
    }
}

float chatblas_snrm2(int n, float *x) {
    float *d_x, *d_res;
    float h_res = 0.0;
    hipMalloc((void**)&d_x, n * sizeof(float));
    hipMalloc((void**)&d_res, sizeof(float));
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_res, &h_res, sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    snrm2_kernel<<<blocksPerGrid, threadsPerBlock>>>(n, d_x, d_res);

    hipMemcpy(&h_res, d_res, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_x);
    hipFree(d_res);

    return sqrtf(h_res);
}