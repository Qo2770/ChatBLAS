#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sdot_kernel(int n, float *x, float *y, float *res) {
    extern __shared__ float shared_data[];
    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + tid;
    float temp_sum = 0.0f;

    if (index < n) {
        temp_sum = x[index] * y[index];
    }

    shared_data[tid] = temp_sum;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_data[tid] += shared_data[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0) {
        atomicAdd(res, shared_data[0]);
    }
}

float chatblas_sdot(int n, float *x, float *y) {
    float *d_x, *d_y, *d_res;
    float result = 0.0f;
    hipMalloc((void**)&d_x, n * sizeof(float));
    hipMalloc((void**)&d_y, n * sizeof(float));
    hipMalloc((void**)&d_res, sizeof(float));
    
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(d_res, 0, sizeof(float));

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    sdot_kernel<<<numBlocks, blockSize, blockSize * sizeof(float)>>>(n, d_x, d_y, d_res);

    hipMemcpy(&result, d_res, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_res);

    return result;
}