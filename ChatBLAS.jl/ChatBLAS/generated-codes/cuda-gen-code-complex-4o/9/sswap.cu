#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sswap_kernel(int n, float *x, float *y) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float temp = x[idx];
        x[idx] = y[idx];
        y[idx] = temp;
    }
}

void chatblas_sswap(int n, float *x, float *y) {
    float *d_x, *d_y;
    size_t size = n * sizeof(float);

    // Allocate device memory
    hipMalloc((void **)&d_x, size);
    hipMalloc((void **)&d_y, size);

    // Copy vectors from host to device
    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

    // Define block and grid sizes
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    // Launch kernel
    sswap_kernel<<<blocksPerGrid, threadsPerBlock>>>(n, d_x, d_y);

    // Copy result back to host
    hipMemcpy(x, d_x, size, hipMemcpyDeviceToHost);
    hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_x);
    hipFree(d_y);
}