#include "hip/hip_runtime.h"
#include "chatblas_cuda.h" 
__global__ void sasum_kernel(int n, float *x, float *sum) { 
	__shared__ float cache[256]; 
	int i = blockIdx.x * blockDim.x + threadIdx.x; 
	int stride = blockDim.x * gridDim.x; 
	float local_sum = 0.0f; 
	for (int j = i; j < n; j += stride) { 
		local_sum += fabsf(x[j]); 
	} 
	cache[threadIdx.x] = local_sum; 
	__syncthreads(); 
	int k = blockDim.x/2; 
	while (k != 0) { 
		if (threadIdx.x < k) { 
			cache[threadIdx.x] += cache[threadIdx.x + k]; 
		} 
		__syncthreads(); 
		k /= 2; 
	} 
	if (threadIdx.x == 0) { 
		atomicAdd(sum, cache[0]); 
	} 
} 

float chatblas_sasum(int n, float *x) { 
	float *d_x, *d_sum; 
	float sum = 0.0f; 
	hipMalloc(&d_x, n * sizeof(float)); 
	hipMalloc(&d_sum, sizeof(float)); 
	hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice); 
	hipMemcpy(d_sum, &sum, sizeof(float), hipMemcpyHostToDevice); 
	int blockSize = 256; 
	int numBlocks = (n + blockSize - 1) / blockSize; 
	sasum_kernel<<<numBlocks, blockSize>>>(n, d_x, d_sum); 
	hipMemcpy(&sum, d_sum, sizeof(float), hipMemcpyDeviceToHost); 
	hipFree(d_x); 
	hipFree(d_sum); 
	return sum; 
}
