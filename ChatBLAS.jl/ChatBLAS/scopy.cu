#include "hip/hip_runtime.h"
#include "chatblas_cuda.h" 

__global__ void scopy_kernel(int n, float *x, float *y) { int index = blockIdx.x * blockDim.x + threadIdx.x; int stride = blockDim.x * gridDim.x; for (int i = index; i < n; i += stride) { y[i] = x[i]; } } 

void chatblas_scopy(int n, float *x, float *y) { float *d_x, *d_y; int size = n * sizeof(float); hipMalloc((void **)&d_x, size); hipMalloc((void **)&d_y, size); hipMemcpy(d_x, x, size, hipMemcpyHostToDevice); int blockSize = 256; int numBlocks = (n + blockSize - 1) / blockSize; scopy_kernel<<<numBlocks, blockSize>>>(n, d_x, d_y); hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost); hipFree(d_x); hipFree(d_y); }
