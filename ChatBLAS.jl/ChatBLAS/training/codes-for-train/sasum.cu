#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sasum_kernel(int n, float *x, float *sum) {
   int index = blockIdx.x * blockDim.x + threadIdx.x;
   int stride = blockDim.x * gridDim.x;

   float local_sum = 0.0f;
   for (int i = index; i < n; i += stride) {
      local_sum += fabsf(x[i]);
   }

   atomicAdd(sum, local_sum);
}

float chatblas_sasum(int n, float *x) {
   float *d_x, *d_sum;
   float sum = 0.0f;

   hipMalloc((void **)&d_x, n * sizeof(float));
   hipMalloc((void **)&d_sum, sizeof(float));

   hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(d_sum, &sum, sizeof(float), hipMemcpyHostToDevice);

   int blockSize = 256;
   int numBlocks = (n + blockSize - 1) / blockSize;

   sasum_kernel<<<numBlocks, blockSize>>>(n, d_x, d_sum);

   hipMemcpy(&sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);

   hipFree(d_x);
   hipFree(d_sum);

   return sum;
}
