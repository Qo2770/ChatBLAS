#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sdot_kernel(int n, float *x, float *y, float *res) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    float result = 0;
    if (index < n) {
        result = x[index] * y[index];
    }

    atomicAdd(res, result);
}

float chatblas_sdot( int n, float *x, float *y) {
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;
    float *d_x, *d_y, *d_res;
    float res = 0;

    hipMalloc(&d_x, n*sizeof(float));
    hipMemcpy(d_x, x, n*sizeof(float), hipMemcpyHostToDevice);

    hipMalloc(&d_y, n*sizeof(float));
    hipMemcpy(d_y, y, n*sizeof(float), hipMemcpyHostToDevice);

    hipMalloc(&d_res, sizeof(float));
    hipMemcpy(d_res, &res, sizeof(float), hipMemcpyHostToDevice);

    sdot_kernel<<<gridSize, blockSize>>>(n, d_x, d_y, d_res);

    hipMemcpy(&res, d_res, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_res);

    return res;
}