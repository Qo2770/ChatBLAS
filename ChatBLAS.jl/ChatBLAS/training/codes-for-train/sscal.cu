#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sscal_kernel(int n, float a, float *x) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (tid < n) {
        x[tid] *= a;
    }
}

void chatblas_sscal(int n, float a, float *x) {
    float *d_x;
    
    hipMalloc((void**)&d_x, n * sizeof(float));
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;
    
    sscal_kernel<<<gridSize, blockSize>>>(n, a, d_x);
    
    hipMemcpy(x, d_x, n * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_x);
}
