#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void swapKernel(int n, float *x, float *y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        float temp = x[index];
        x[index] = y[index];
        y[index] = temp;
    }
}

void chatblas_sswap(int n, float *x, float *y) {
    // Allocate device memory
    float *dev_x, *dev_y;
    hipMalloc(&dev_x, n * sizeof(float));
    hipMalloc(&dev_y, n * sizeof(float));
    
    // Copy input vectors from host to device memory
    hipMemcpy(dev_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_y, y, n * sizeof(float), hipMemcpyHostToDevice);
    
    // Perform swap on device
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;
    swapKernel<<<gridSize, blockSize>>>(n, dev_x, dev_y);
    
    // Copy results back from device to host memory
    hipMemcpy(x, dev_x, n * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(y, dev_y, n * sizeof(float), hipMemcpyDeviceToHost);
    
    // Free device memory
    hipFree(dev_x);
    hipFree(dev_y);
}
