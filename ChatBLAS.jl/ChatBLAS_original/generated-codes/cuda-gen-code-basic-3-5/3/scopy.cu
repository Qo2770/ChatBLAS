#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void copy_kernel(int n, float *x, float *y) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
        y[idx] = x[idx];
}

void chatblas_scopy(int n, float *x, float *y) {
    // Allocate memory on the GPU
    float *d_x, *d_y;
    hipMalloc((void **)&d_x, n * sizeof(float));
    hipMalloc((void **)&d_y, n * sizeof(float));

    // Copy input data from host to device
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);

    // Invoke kernel on the GPU
    int threads_per_block = 256;
    int num_blocks = (n + threads_per_block - 1) / threads_per_block;
    copy_kernel<<<num_blocks, threads_per_block>>>(n, d_x, d_y);

    // Copy result back from device to host
    hipMemcpy(y, d_y, n * sizeof(float), hipMemcpyDeviceToHost);

    // Free the GPU memory
    hipFree(d_x);
    hipFree(d_y);
}
