#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sscal_kernel(int n, float a, float *x) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < n) {
        x[i] *= a;
    }
}

void chatblas_sscal(int n, float a, float *x) {
    const int blockSize = 256;
    const int numBlocks = (n + blockSize - 1) / blockSize;
    
    float *d_x;
    hipMalloc((void **)&d_x, n * sizeof(float));
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    
    sscal_kernel<<<numBlocks, blockSize>>>(n, a, d_x);
    
    hipMemcpy(x, d_x, n * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_x);
}
