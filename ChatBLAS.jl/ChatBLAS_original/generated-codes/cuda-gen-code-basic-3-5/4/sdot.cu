#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void dotProduct(int n, float *x, float *y, float *result) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    float partialSum = 0.0f;
    for (int i = tid; i < n; i += stride) {
        partialSum += x[i] * y[i];
    }

    atomicAdd(result, partialSum);
}

float chatblas_sdot(int n, float *x, float *y) {
    float *deviceX, *deviceY, *deviceResult;
    float result = 0.0f;

    hipMalloc((void **)&deviceX, n * sizeof(float));
    hipMalloc((void **)&deviceY, n * sizeof(float));
    hipMalloc((void **)&deviceResult, sizeof(float));

    hipMemcpy(deviceX, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceY, y, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(deviceResult, 0, sizeof(float));

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    dotProduct<<<numBlocks, blockSize>>>(n, deviceX, deviceY, deviceResult);

    hipMemcpy(&result, deviceResult, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(deviceX);
    hipFree(deviceY);
    hipFree(deviceResult);

    return result;
}
