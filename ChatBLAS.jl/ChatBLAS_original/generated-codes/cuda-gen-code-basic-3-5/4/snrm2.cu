#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void computeNorm(float *x, float *result, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        atomicAdd(result, x[idx] * x[idx]);
    }
}

float chatblas_snrm2(int n, float *x) {
    float *d_x, *d_result;
    float result = 0.0f;
    hipMalloc(&d_x, n * sizeof(float));
    hipMalloc(&d_result, sizeof(float));
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;
    computeNorm<<<gridSize, blockSize>>>(d_x, d_result, n);
    
    hipMemcpy(&result, d_result, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_x);
    hipFree(d_result);
    
    return sqrt(result);
}
