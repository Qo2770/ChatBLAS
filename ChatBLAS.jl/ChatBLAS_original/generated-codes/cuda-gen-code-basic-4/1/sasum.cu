#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"
#include <stdio.h>
#include <math.h>

// Kerel function to compute absolute sum
__global__ void abs_sum(const float *x, float *y, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
        atomicAdd(y, abs(x[i]));
}

// ChatBlas SASUM function
float chatblas_sasum(int n, float *x)
{
    float *dev_x, *dev_y;
    float y = 0.0;

    // Move data to device
    hipMalloc((void **)&dev_x, n * sizeof(float));
    hipMemcpy(dev_x, x, n * sizeof(float), hipMemcpyHostToDevice);

    hipMalloc((void **)&dev_y, sizeof(float));
    hipMemcpy(dev_y, &y, sizeof(float), hipMemcpyHostToDevice);

    // Configure and launch the kernel
    dim3 dimBlock(256);
    dim3 dimGrid((n + dimBlock.x - 1)/dimBlock.x);

    abs_sum<<<dimGrid, dimBlock>>>(dev_x, dev_y, n);

    // Move result back to host
    hipMemcpy(&y, dev_y, sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(dev_x);
    hipFree(dev_y);

    return y;
}
