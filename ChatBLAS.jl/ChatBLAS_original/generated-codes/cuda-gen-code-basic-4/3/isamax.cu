#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void findMaxIndexKernel(float* d_x, int* d_maxIndex, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    float max_val;

    if(i < n)
    {
        if (fabs(d_x[i]) > max_val)
        {
            max_val = fabs(d_x[i]);
            *d_maxIndex = i;
        }
    }
}


int chatblas_isamax(int n, float *x){
   
    float* d_x;
    int* d_maxIndex;
    int maxIndex = 0;

    hipMalloc((void**)&d_x, n * sizeof(float));
    hipMalloc((void**)&d_maxIndex, sizeof(int));

    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_maxIndex, &maxIndex, sizeof(int), hipMemcpyHostToDevice);

    findMaxIndexKernel<<<(n+255)/256, 256>>>(d_x, d_maxIndex, n);

    hipMemcpy(&maxIndex, d_maxIndex, sizeof(int), hipMemcpyDeviceToHost);
    
    hipFree(d_x);
    hipFree(d_maxIndex);
        
    return maxIndex;
}
