#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"
#include <math.h>
__global__ void sum_abs_kernel(float* x, float* y, unsigned int n) {
    unsigned int index = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    __shared__ float cache[256];

    float temp = 0;
    while(index < n) {
        temp += fabs(x[index]);
        index += stride;
    }

    cache[threadIdx.x] = temp;

    __syncthreads();

    if(threadIdx.x == 0) {
        float sum = 0;
        for(int i = 0; i < blockDim.x; i++)
            sum += cache[i];
        
        atomicAdd(y, sum);
    }
}

float chatblas_sasum(int n, float *x) {
    float *dev_x, *dev_y;
    float result;

    hipMalloc((void**)&dev_x, n*sizeof(float));
    hipMalloc((void**)&dev_y, sizeof(float));

    hipMemcpy(dev_x, x, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemset(dev_y, 0, sizeof(float));

    sum_abs_kernel<<<256,256>>>(dev_x, dev_y, n);

    hipMemcpy(&result, dev_y, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(dev_x);
    hipFree(dev_y);

    return result;
}
