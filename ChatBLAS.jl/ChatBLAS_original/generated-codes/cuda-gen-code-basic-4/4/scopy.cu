#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void chatblas_scopy_device(int n, float *x, float *y) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < n) {
        y[index] = x[index];
    }
}

extern "C" {
    void chatblas_scopy(int n, float *x, float *y) {
        // compute number of blocks needed
        int blocks = (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

        // allocate device memory
        float *d_x, *d_y;
        hipMalloc((void**) &d_x, sizeof(float) * n);
        hipMalloc((void**) &d_y, sizeof(float) * n);

        // copy input to device
        hipMemcpy(d_x, x, sizeof(float) * n, hipMemcpyHostToDevice);

        // Execute copy kernel
        chatblas_scopy_device<<<blocks, THREADS_PER_BLOCK>>>(n, d_x, d_y);
        
        // copy result back to host
        hipMemcpy(y, d_y, sizeof(float) * n, hipMemcpyDeviceToHost);

        // release device memory
        hipFree(d_x);
        hipFree(d_y);
    }
}
