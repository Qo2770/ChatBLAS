#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void kernel_sdsdot(int n, float *x, float *y, float *z) {
   int index = threadIdx.x + blockDim.x * blockIdx.x;
   int stride = blockDim.x * gridDim.x;
   for (int i = index; i < n; i += stride)
       z[i] = ((double) x[i]) * ((double) y[i]);
}

extern "C"
float chatblas_sdsdot(int n, float b, float *x, float *y) {
   int size = n * sizeof(float);
   float *d_x, *d_y, *d_z, result = 0.0, *partial_sums;
   
   hipMalloc(&d_x, size);
   hipMalloc(&d_y, size);
   hipMalloc(&d_z, size);
   hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
   hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);
   
   int threads_per_block = 256;
   int blocks_per_grid =(n + threads_per_block - 1) / threads_per_block;

   kernel_sdsdot<<<blocks_per_grid, threads_per_block>>>(n, d_x, d_y, d_z);

   partial_sums = (float*)malloc(blocks_per_grid*sizeof(float));
   hipMemcpy(partial_sums, d_z, blocks_per_grid*sizeof(float), hipMemcpyDeviceToHost);

   for(int i = 0; i < blocks_per_grid ; i++)
       result += partial_sums[i];
   
   hipFree(d_x);
   hipFree(d_y);
   hipFree(d_z);
   free(partial_sums);
   
   result += b;
   return result;
}
