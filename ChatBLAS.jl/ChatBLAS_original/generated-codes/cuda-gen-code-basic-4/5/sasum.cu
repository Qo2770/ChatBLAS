#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"
 
// CUDA Kernel function to add the absolute values of the elements
__global__ void addAbs(int n, float *x, float *result) {
    int index = threadIdx.x;
    int stride = blockDim.x;
   
    for (int i = index; i < n; i += stride){
        atomicAdd(result, fabsf(x[i]));
    }
}

float chatblas_sasum(int n, float *x) {
    float *d_x, *d_result;
    float result = 0.0;

    // Allocate device memory
    hipMalloc(&d_x, n*sizeof(float));
    hipMalloc(&d_result, sizeof(float));

    // Copy vectors from host to device memory
    hipMemcpy(d_x, x, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_result, &result, sizeof(float), hipMemcpyHostToDevice);

    // Launch CUDA Kernel
    addAbs<<<1, 256>>>(n, d_x, d_result);

    // Copy result back to host memory
    hipMemcpy(&result, d_result, sizeof(float), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_result);
    hipFree(d_x);

    return result;
}
