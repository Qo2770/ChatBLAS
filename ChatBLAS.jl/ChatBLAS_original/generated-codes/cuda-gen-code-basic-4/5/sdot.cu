#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include "chatblas_cuda.h"

__global__ void dotProductKernel(int n, float *x, float *y, float *result) {
    extern __shared__ float temp[];

	int index = threadIdx.x;
	int stride = blockDim.x;

	temp[index] = 0;

	if (index < n) {
		for (int i = index; i < n; i += stride)
			temp[index] += x[i] * y[i];
	}

	__syncthreads();

	// Use CUB's DeviceReduce to get the final dot product result
	hipcub::DeviceReduce::Sum(temp, result, n);
}

float chatblas_sdot(int n, float *x, float *y) {
    float *d_x, *d_y, *d_result;
    float result;

    // allocate memory on the device
    hipMalloc(&d_x, n * sizeof(float));
    hipMalloc(&d_y, n * sizeof(float));
    hipMalloc(&d_result, sizeof(float));

    // copy inputs to device
    hipMemcpy(d_x, x, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n*sizeof(float), hipMemcpyHostToDevice);

    // launch kernel
    dotProductKernel<<<1, 256, 256*sizeof(float)>>>(n, d_x, d_y, d_result);

    // copy result back to host
    hipMemcpy(&result, d_result, sizeof(float), hipMemcpyDeviceToHost);

    // cleanup
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_result);

    return result;
}
