#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void saxpy_kernel(int n, float a, float *x, float *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        y[i] = a * x[i] + y[i];
    }
}

void chatblas_saxpy(int n, float a, float *x, float *y) {
    float *d_x, *d_y;
    int size = n * sizeof(float);

    // Allocate GPU memory for vectors
    hipMalloc((void **)&d_x, size);
    hipMalloc((void **)&d_y, size);

    // Transfer data from CPU to GPU
    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

    // Define block and grid size
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    // Call kernel function
    saxpy_kernel<<<blocksPerGrid, threadsPerBlock>>>(n, a, d_x, d_y);

    // Transfer data from GPU to CPU
    hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_x);
    hipFree(d_y);
}