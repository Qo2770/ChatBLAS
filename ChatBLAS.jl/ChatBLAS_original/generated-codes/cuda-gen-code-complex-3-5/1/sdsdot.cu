#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sdsdot_kernel(int n, float b, float *x, float *y, double *res) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    double acc = 0.0;
    while (idx < n) {
        acc += double(x[idx]) * double(y[idx]);
        idx += blockDim.x * gridDim.x;
    }

    atomicAdd(res, acc + double(b));
}

float chatblas_sdsdot(int n, float b, float *x, float *y) {
    // Allocate GPU memory for the vectors
    float *d_x, *d_y;
    hipMalloc((void **)&d_x, n * sizeof(float));
    hipMalloc((void **)&d_y, n * sizeof(float));

    // Copy the vectors from CPU to GPU memory
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);

    // Define block size and number of blocks
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    // Allocate GPU memory for the result
    double *d_res;
    hipMalloc((void **)&d_res, sizeof(double));
    hipMemset(d_res, 0, sizeof(double));

    // Launch the kernel
    sdsdot_kernel<<<numBlocks, blockSize>>>(n, b, d_x, d_y, d_res);

    // Copy the result from GPU to CPU memory
    double res;
    hipMemcpy(&res, d_res, sizeof(double), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_res);

    return float(res);
}