#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sswap_kernel(int n, float *x, float *y) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < n) {
        float temp = x[id];
        x[id] = y[id];
        y[id] = temp;
    }
}

void chatblas_sswap(int n, float *x, float *y) {
    // Allocate GPU memory for x and y
    float *dev_x, *dev_y;
    hipMalloc((void**)&dev_x, n * sizeof(float));
    hipMalloc((void**)&dev_y, n * sizeof(float));
    
    // Copy data from CPU to GPU
    hipMemcpy(dev_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_y, y, n * sizeof(float), hipMemcpyHostToDevice);
    
    // Define block size and number of blocks
    int block_size = 256;
    int num_blocks = (n + block_size - 1) / block_size;
    
    // Call the kernel
    sswap_kernel<<<num_blocks, block_size>>>(n, dev_x, dev_y);
    
    // Copy result from GPU to CPU
    hipMemcpy(x, dev_x, n * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(y, dev_y, n * sizeof(float), hipMemcpyDeviceToHost);
    
    // Free GPU memory
    hipFree(dev_x);
    hipFree(dev_y);
}
