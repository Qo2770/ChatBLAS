#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sasum_kernel(int n, float *x, float *sum) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    float local_sum = 0.0f;

    for (int i = tid; i < n; i += stride) {
        local_sum += fabsf(x[i]);
    }

    atomicAdd(sum, local_sum);
}

float chatblas_sasum(int n, float *x) {
    float *d_x, *d_sum, sum;

    // Allocate memory for GPU vectors
    hipMalloc((void**)&d_x, n * sizeof(float));
    hipMalloc((void**)&d_sum, sizeof(float));

    // Copy input vector from CPU to GPU
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);

    // Set initial sum to 0.0
    sum = 0.0f;
    hipMemcpy(d_sum, &sum, sizeof(float), hipMemcpyHostToDevice);

    // Define the number of threads per block and the number of blocks
    int threadsPerBlock = 256;
    int numBlocks = (n + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel
    sasum_kernel<<<numBlocks, threadsPerBlock>>>(n, d_x, d_sum);

    // Copy the sum from GPU to CPU
    hipMemcpy(&sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_x);
    hipFree(d_sum);

    return sum;
}