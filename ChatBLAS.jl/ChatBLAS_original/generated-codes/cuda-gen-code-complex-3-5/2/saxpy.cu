#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void saxpy_kernel(int n, float a, float *x, float *y) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < n) {
    y[index] = a * x[index] + y[index];
  }
}

void chatblas_saxpy(int n, float a, float *x, float *y) {
  // Allocate GPU memory for x and y
  float *d_x, *d_y;
  hipMalloc((void**)&d_x, n * sizeof(float));
  hipMalloc((void**)&d_y, n * sizeof(float));

  // Transfer x and y from CPU to GPU
  hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);

  // Set the number of blocks and threads per block
  int threadsPerBlock = 256;
  int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

  // Launch the kernel
  saxpy_kernel<<<blocksPerGrid, threadsPerBlock>>>(n, a, d_x, d_y);

  // Transfer the result from GPU to CPU
  hipMemcpy(y, d_y, n * sizeof(float), hipMemcpyDeviceToHost);

  // Free GPU memory
  hipFree(d_x);
  hipFree(d_y);
}