#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void scopy_kernel(int n, float *x, float *y) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (tid < n) {
        y[tid] = x[tid];
    }
}

void chatblas_scopy(int n, float *x, float *y) {
    float *d_x, *d_y;

    // Allocate GPU memory for x and y
    hipMalloc((void**)&d_x, n * sizeof(float));
    hipMalloc((void**)&d_y, n * sizeof(float));

    // Copy data from CPU to GPU
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);

    // Define block size and number of blocks
    int blocksize = 256;
    int numBlocks = (n + blocksize - 1) / blocksize;

    // Launch the kernel
    scopy_kernel<<<numBlocks, blocksize>>>(n, d_x, d_y);

    // Copy data back from GPU to CPU
    hipMemcpy(y, d_y, n * sizeof(float), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_x);
    hipFree(d_y);
}
