#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void snrm2_kernel(int n, float *x, float *res) {
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    float sum = 0.0f;
    for (int i = thread_id; i < n; i += stride) {
        sum += x[i] * x[i];
    }

    __shared__ float shared_sum[256];
    shared_sum[threadIdx.x] = sum;
    __syncthreads();

    int i = blockDim.x / 2;
    while (i != 0) {
        if (threadIdx.x < i) {
            shared_sum[threadIdx.x] += shared_sum[threadIdx.x + i];
        }
        __syncthreads();
        i /= 2;
    }

    if (threadIdx.x == 0) {
        atomicAdd(res, shared_sum[0]);
    }
}

float chatblas_snrm2(int n, float *x) {
    float *gpu_x, *gpu_res;

    hipMalloc((void **)&gpu_x, n * sizeof(float));
    hipMemcpy(gpu_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc((void **)&gpu_res, sizeof(float));
    hipMemset(gpu_res, 0, sizeof(float));

    int threads_per_block = 256;
    int blocks_per_grid = (n + threads_per_block - 1) / threads_per_block;

    snrm2_kernel<<<blocks_per_grid, threads_per_block>>>(n, gpu_x, gpu_res);
    hipDeviceSynchronize();

    float res;
    hipMemcpy(&res, gpu_res, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(gpu_x);
    hipFree(gpu_res);

    return sqrt(res);
}
