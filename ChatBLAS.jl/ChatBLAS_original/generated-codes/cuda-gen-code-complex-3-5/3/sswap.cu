#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sswap_kernel(int n, float *x, float *y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < n){
        float temp = x[index];
        x[index] = y[index];
        y[index] = temp;
    }
}

void chatblas_sswap(int n, float *x, float *y) {
    float *d_x, *d_y;
    int size = n * sizeof(float);

    hipMalloc((void **)&d_x, size);
    hipMalloc((void **)&d_y, size);

    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    sswap_kernel<<<blocksPerGrid, threadsPerBlock>>>(n, d_x, d_y);

    hipMemcpy(x, d_x, size, hipMemcpyDeviceToHost);
    hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
}