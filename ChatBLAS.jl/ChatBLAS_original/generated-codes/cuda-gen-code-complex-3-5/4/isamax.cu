#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void isamax_kernel(int n, float *x, float *ind) {
  // Get the thread ID
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  
  // Initialize variables
  float max_val = 0.0f;
  int max_idx = 0;
  
  // Find the element with largest absolute value
  for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
    float abs_val = abs(x[i]);
    if (abs_val > max_val) {
      max_val = abs_val;
      max_idx = i;
    }
  }
  
  // Store the result in the global memory
  if (tid == 0)
    *ind = (float)max_idx;
}

int chatblas_isamax(int n, float *x) {
  // Declare device vectors
  float *d_x, *d_ind;
  
  // Allocate memory on the GPU
  hipMalloc((void**)&d_x, n * sizeof(float));
  hipMalloc((void**)&d_ind, sizeof(float));
  
  // Copy input vector from host to device
  hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
  
  // Calculate the number of threads and blocks
  int threadsPerBlock = 256;
  int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
  
  // Launch the kernel
  isamax_kernel<<<blocksPerGrid, threadsPerBlock>>>(n, d_x, d_ind);
  
  // Copy the result back from device to host
  float max_idx;
  hipMemcpy(&max_idx, d_ind, sizeof(float), hipMemcpyDeviceToHost);
  
  // Free the GPU memory
  hipFree(d_x);
  hipFree(d_ind);
  
  // Return the position of the largest absolute value
  return (int)max_idx;
}
