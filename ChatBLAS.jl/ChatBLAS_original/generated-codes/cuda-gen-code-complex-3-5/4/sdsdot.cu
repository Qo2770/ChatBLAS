#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sdsdot_kernel(int n, float b, float *x, float *y, float *res) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    double sum = 0.0;
    for (int i = tid; i < n; i += stride) {
        sum += (double)(x[i]) * (double)(y[i]);
    }

    atomicAdd(res, sum + (double)b);
}

float chatblas_sdsdot(int n, float b, float *x, float *y) {
    float *d_x, *d_y, *d_res;
    float result;

    hipMalloc((void **)&d_x, n * sizeof(float));
    hipMalloc((void **)&d_y, n * sizeof(float));
    hipMalloc((void **)&d_res, sizeof(float));

    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    sdsdot_kernel<<<numBlocks, blockSize>>>(n, b, d_x, d_y, d_res);

    hipMemcpy(&result, d_res, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_res);

    return result;
}