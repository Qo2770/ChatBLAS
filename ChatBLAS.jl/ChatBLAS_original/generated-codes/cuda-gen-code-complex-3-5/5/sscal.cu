#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sscal_kernel(int n, float a, float *x) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < n) {
    x[tid] *= a;
  }
}

void chatblas_sscal(int n, float a, float *x) {
  float *d_x;
  int size = n * sizeof(float);

  // allocate memory on GPU
  hipMalloc((void **)&d_x, size);

  // copy input data from CPU to GPU
  hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);

  // define the number of blocks of threads and the size of each block
  int threadsPerBlock = 256;
  int numBlocks = (n + threadsPerBlock - 1) / threadsPerBlock;

  // call the kernel function
  sscal_kernel<<<numBlocks, threadsPerBlock>>>(n, a, d_x);

  // copy the result back from GPU to CPU
  hipMemcpy(x, d_x, size, hipMemcpyDeviceToHost);

  // free the GPU memory
  hipFree(d_x);
}
