#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sdsdot_kernel(int n, float b, float *x, float *y, float *res) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n) {
        res[tid] = static_cast<double>(x[tid]) * static_cast<double>(y[tid]);
    }
}

float chatblas_sdsdot(int n, float b, float *x, float *y) {
    float *d_x, *d_y, *d_res;
    float result = 0.0;

    hipMalloc((void**)&d_x, n * sizeof(float));
    hipMalloc((void**)&d_y, n * sizeof(float));
    hipMalloc((void**)&d_res, n * sizeof(float));

    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    sdsdot_kernel<<<numBlocks, blockSize>>>(n, b, d_x, d_y, d_res);

    hipMemcpy(&result, d_res, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_res);

    return result + b;
}

