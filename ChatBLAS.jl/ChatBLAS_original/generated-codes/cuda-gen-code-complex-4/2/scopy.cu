#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void scopy_kernel( int n, float *x, float *y ) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n) {
        y[idx] = x[idx];
    }
}

void chatblas_scopy(int n, float *x, float *y) {
    float *x_gpu, *y_gpu;

    hipMalloc((void**) &x_gpu, n*sizeof(float));
    hipMalloc((void**) &y_gpu, n*sizeof(float));

    hipMemcpy(x_gpu, x, n*sizeof(float), hipMemcpyHostToDevice);

    int blocks = (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    scopy_kernel<<<blocks, THREADS_PER_BLOCK>>>(n, x_gpu, y_gpu);

    hipMemcpy(y, y_gpu, n*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(x_gpu);
    hipFree(y_gpu);
}