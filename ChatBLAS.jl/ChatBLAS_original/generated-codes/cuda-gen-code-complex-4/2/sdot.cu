#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sdot_kernel(int n, float *x, float *y, float *res) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    __shared__ float temp_res[256];
    temp_res[threadIdx.x] = (index < n)? x[index] * y[index] : 0;

    __syncthreads();

    if (threadIdx.x == 0) {
        float sum = 0;
        for (int i = 0; i < 256; i++) {
            sum += temp_res[i];
        }
        atomicAdd(res, sum);
    }
}

float chatblas_sdot(int n, float *x, float *y) {
    float *dev_x = 0;
    float *dev_y = 0;
    float *dev_res = 0;
    float res = 0;

    hipMalloc((void**)&dev_x, n * sizeof(float));
    hipMalloc((void**)&dev_y, n * sizeof(float));
    hipMalloc((void**)&dev_res, sizeof(float));

    hipMemcpy(dev_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_y, y, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_res, &res, sizeof(float), hipMemcpyHostToDevice);

    int blocks = (n + 255) / 256;
    sdot_kernel <<< blocks, 256 >>> (n, dev_x, dev_y, dev_res);

    hipMemcpy(&res, dev_res, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(dev_x);
    hipFree(dev_y);
    hipFree(dev_res);

    return res;
}