#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void sswap_kernel(int n, float *x, float *y) {
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  float temp;

  if (index < n) {
    temp = x[index];
    x[index] = y[index];
    y[index] = temp;
  }
}

void chatblas_sswap(int n, float *x, float *y) {
  float *device_x, *device_y;

  hipMalloc(&device_x, n*sizeof(float));
  hipMalloc(&device_y, n*sizeof(float));

  hipMemcpy(device_x, x, n*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(device_y, y, n*sizeof(float), hipMemcpyHostToDevice);

  int blockSize = 256;
  int numBlocks = (n + blockSize - 1) / blockSize;
  sswap_kernel<<<numBlocks, blockSize>>>(n, device_x, device_y);

  hipMemcpy(x, device_x, n*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(y, device_y, n*sizeof(float), hipMemcpyDeviceToHost);

  hipFree(device_x);
  hipFree(device_y);
}