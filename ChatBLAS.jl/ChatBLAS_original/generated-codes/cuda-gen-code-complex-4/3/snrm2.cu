#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void snrm2_kernel(int n, float *x, float *res) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        atomicAdd(res, x[idx] * x[idx]);
    }
}

float chatblas_snrm2(int n, float *x) {
    const int blocks = (n + 255) / 256;
    const int threads = min(n, 256);
    float *d_x, *d_res, h_res = 0;
    hipMalloc((void **)&d_x, n * sizeof(float));
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc((void **)&d_res, sizeof(float));
    hipMemcpy(d_res, &h_res, sizeof(float), hipMemcpyHostToDevice);
    snrm2_kernel<<<blocks, threads>>>(n, d_x, d_res);
    hipMemcpy(&h_res, d_res, sizeof(float), hipMemcpyDeviceToHost);
    h_res = sqrt(h_res);
    hipFree(d_x);
    hipFree(d_res);
    return h_res;
}