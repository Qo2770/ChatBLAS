#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void scopy_kernel(int n, float *x, float *y) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        y[index] = x[index];
    }
}

void chatblas_scopy(int n, float *x, float *y) {

    float *d_x, *d_y;
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;

    hipMalloc((void**)&d_x, n * sizeof(float)); 
    hipMalloc((void**)&d_y, n * sizeof(float));

    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);

    scopy_kernel<<<gridSize, blockSize>>>(n, d_x, d_y);

    hipMemcpy(y, d_y, n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
}