#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void snrm2_kernel(int n, float *x, float *res) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    float temp = 0.0;
    if (idx < n) {
        temp = x[idx];
    }
    __syncthreads();

    atomicAdd(res, temp * temp);
    __syncthreads();
}

float chatblas_snrm2(int n, float *x) {
    float *x_d, *res, result;

    hipMalloc(&x_d, sizeof(float) * n);
    hipMalloc(&res, sizeof(float));

    hipMemcpy(x_d, x, sizeof(float) * n, hipMemcpyHostToDevice);

    float initial_value = 0.0;
    hipMemcpy(res, &initial_value, sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    snrm2_kernel <<<blocksPerGrid, threadsPerBlock>>> (n, x_d, res);

    hipMemcpy(&result, res, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(x_d);
    hipFree(res);

    return sqrt(result);
}