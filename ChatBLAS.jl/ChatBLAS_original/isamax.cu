#include "hip/hip_runtime.h"
#include "chatblas_cuda.h" 
__global__ void isamax_kernel(int n, float *x, float *ind) { int i = blockDim.x * blockIdx.x + threadIdx.x; if (i < n) { if (abs(x[i]) > abs(x[(int)(*ind)])) { *ind = (float)i; } } } 

int chatblas_isamax(int n, float *x) { float *x_d, *ind_d; float ind_h = 0; hipMalloc((void **)&x_d, n * sizeof(float)); hipMalloc((void **)&ind_d, sizeof(float)); hipMemcpy(x_d, x, n * sizeof(float), hipMemcpyHostToDevice); hipMemcpy(ind_d, &ind_h, sizeof(float), hipMemcpyHostToDevice); int blockSize = 256; int numBlocks = (n + blockSize - 1) / blockSize; isamax_kernel<<<numBlocks, blockSize>>>(n, x_d, ind_d); hipMemcpy(&ind_h, ind_d, sizeof(float), hipMemcpyDeviceToHost); hipFree(x_d); hipFree(ind_d); return (int) ind_h; }
