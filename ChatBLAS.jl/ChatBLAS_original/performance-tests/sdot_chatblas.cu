#include <sys/time.h>
#include <hipblas.h>
#include "chatblas_cuda.h"

int main()
{

  struct timeval stop, start;    
  float *host_x, *host_y;
  float *dev_x, *dev_y, *dev_result;
  float result_cublas, result_chatblas;
  hipblasHandle_t h;

  int N = 500000000;
  host_x = (float *) malloc(N * sizeof(float));
  host_y = (float *) malloc(N * sizeof(float));
  
  for (int i=0; i<N; ++i) {
    host_x[i] = 1.0;
    host_y[i] = 1.0;
  }
  
 
  hipblasCreate(&h);
  //hipblasSetPointerMode(h, HIPBLAS_POINTER_MODE_DEVICE);

  gettimeofday(&start, NULL);
  
  hipMalloc( (void**)&dev_x, N*sizeof(float));
  hipMalloc( (void**)&dev_y, N*sizeof(float));
  hipMalloc( (void**)&dev_result, sizeof(float));

  hipblasSetVector(N, sizeof(host_x[0]), host_x, 1, dev_x, 1);
  hipblasSetVector(N, sizeof(host_y[0]), host_y, 1, dev_y, 1);
  hipDeviceSynchronize();

  hipblasSdot(h, N, dev_x, 1, dev_y, 1, dev_result);
  hipDeviceSynchronize();

  hipMemcpy(&result_cublas, dev_result, sizeof(float), hipMemcpyDeviceToHost);

  gettimeofday(&stop, NULL);
  printf("cuBLAS took %lu us\n", (stop.tv_sec - start.tv_sec) * 1000000 + stop.tv_usec - start.tv_usec);

  if (h)
    hipblasDestroy(h);
  hipFree(dev_y);
  hipFree(dev_x);

  gettimeofday(&start, NULL);

  result_chatblas = chatblas_sdot( N, host_x, host_y);

  gettimeofday(&stop, NULL);
  printf("chatBLAS took %lu us\n", (stop.tv_sec - start.tv_sec) * 1000000 + stop.tv_usec - start.tv_usec);

  if(result_chatblas != result_cublas){
    printf("Error, cuBLAS = %2.f, chatBLAS = %2.f\n", result_cublas, result_chatblas);
  }
	    
	  
  free(host_y);
  free(host_x);

  return 0;
}



