#include "hip/hip_runtime.h"
#include "chatblas_cuda.h" 

__global__ void snrm2_kernel(int n, float *x, float *res) { int index = blockIdx.x * blockDim.x + threadIdx.x; int stride = blockDim.x * gridDim.x; float local_res = 0.0f; for (int i = index; i < n; i += stride) { local_res += x[i] * x[i]; } atomicAdd(res, local_res); } 

float chatblas_snrm2(int n, float *x) { int blockSize = 256; int numBlocks = (n + blockSize - 1) / blockSize; float *d_x, *d_res; float res = 0.0f; hipMalloc((void **)&d_x, n * sizeof(float)); hipMalloc((void **)&d_res, sizeof(float)); hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice); hipMemcpy(d_res, &res, sizeof(float), hipMemcpyHostToDevice); snrm2_kernel<<<numBlocks, blockSize>>>(n, d_x, d_res); hipMemcpy(&res, d_res, sizeof(float), hipMemcpyDeviceToHost); res = sqrt(res); hipFree(d_x); hipFree(d_res); return res; }
