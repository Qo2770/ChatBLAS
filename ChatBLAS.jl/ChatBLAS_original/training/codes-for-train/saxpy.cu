#include "hip/hip_runtime.h"
#include "chatblas_cuda.h"

__global__ void saxpy_kernel(int n, float a, float *x, float *y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (index < n)
    {
        y[index] = a * x[index] + y[index];
    }
}

void chatblas_saxpy(int n, float a, float *x, float *y)
{
    int size = n * sizeof(float);
    float *d_x, *d_y;
    
    hipMalloc((void **)&d_x, size);
    hipMalloc((void **)&d_y, size);
    
    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);
    
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    
    saxpy_kernel<<<blocksPerGrid, threadsPerBlock>>>(n, a, d_x, d_y);

    hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);
    
    hipFree(d_x);
    hipFree(d_y);
}

